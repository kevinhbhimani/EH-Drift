#include "hip/hip_runtime.h"
/*
    Performs sum of charge densities which is needed to calculate signal collected
    author:           Kevin H Bhimani
    first written:    Dec 2021
*/
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "mjd_siggen.h"
#include "detector_geometry.h"
#include "gpu_vars.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__managed__ double grid_sig;
__managed__ double passivated_surface_thickness_sig;
__managed__ double dz_correction;



extern "C" double get_signal_gpu(MJD_Siggen_Setup *setup, GPU_data *gpu_setup, int L, int R, int n_iter, double grid, int save_time, int num_threads);

__global__ void cal_esum1(int L, int R, double *rho_sum, double *rho_e, double *surface_rho_e, double *w_pot, int max_threads){
    // esum1 += rho_e[0][z][r] * (double) (r-1) * setup.wpot[r-1][z-1];
    int r = blockIdx.x%R;
    int z = (floorf(blockIdx.x/R) * max_threads) + threadIdx.x;
    
    if(r==0 || z==0 || r>=R || z>=L){
      return;
    }

    if(z==1){
      rho_sum[((R+1)*z)+r] = rho_e[(0*(L+1)*(R+1))+((R+1)*z)+r] * (double) (r-1) * w_pot[((R+1)*(z-1))+(r-1)] ;
      rho_sum[((R+1)*z)+r] += surface_rho_e[r] * (double) (r-1) * w_pot[((R+1)*(z-1))+(r-1)]*dz_correction;
    }
    else{
      rho_sum[((R+1)*z)+r] = rho_e[(0*(L+1)*(R+1))+((R+1)*z)+r] * (double) (r-1) * w_pot[((R+1)*(z-1))+(r-1)];
    }
  }


__global__ void cal_esum2(int L, int R, double *rho_sum, double *rho_e, double *surface_rho_e, int max_threads){
  int r = blockIdx.x%R;
  int z = (floorf(blockIdx.x/R) * max_threads) + threadIdx.x;
  
  if(r==0 || z==0 || r>=R || z>=L){
    return;
  }
  if(z==1){
    rho_sum[((R+1)*z)+r] = rho_e[(0*(L+1)*(R+1))+((R+1)*z)+r] * (double) (r-1);
    rho_sum[((R+1)*z)+r] += surface_rho_e[r] * (double) (r-1);
  }
  else{
    rho_sum[((R+1)*z)+r] = rho_e[(0*(L+1)*(R+1))+((R+1)*z)+r] * (double) (r-1);
  }
}
__global__ void cal_hsum1(int L, int R, double *rho_sum, double *rho_h, double *surface_rho_h, double *w_pot, int max_threads){
  int r = blockIdx.x%R;
  int z = (floorf(blockIdx.x/R) * max_threads) + threadIdx.x;
  
  if(r==0 || z==0 || r>=R || z>=L){
    return;
  }

  if(z==1){
    rho_sum[((R+1)*z)+r] = rho_h[(0*(L+1)*(R+1))+((R+1)*z)+r] * (double) (r-1) * w_pot[((R+1)*(z-1))+(r-1)];
    rho_sum[((R+1)*z)+r] += surface_rho_h[r] * (double) (r-1) * w_pot[((R+1)*(z-1))+(r-1)]*dz_correction;
  }
  else{
    rho_sum[((R+1)*z)+r] = rho_h[(0*(L+1)*(R+1))+((R+1)*z)+r] * (double) (r-1) * w_pot[((R+1)*(z-1))+(r-1)];
  }
}

__global__ void cal_hsum2(int L, int R, double *rho_sum, double *rho_h, double *surface_rho_h, int max_threads){
  int r = blockIdx.x%R;
  int z = (floorf(blockIdx.x/R) * max_threads) + threadIdx.x;
  
  if(r==0 || z==0 || r>=R || z>=L){
    return;
  }
  if(z==1){
    rho_sum[((R+1)*z)+r] = rho_h[(0*(L+1)*(R+1))+((R+1)*z)+r] * (double) (r-1);
    rho_sum[((R+1)*z)+r] += surface_rho_h[r] * (double) (r-1);
  }
  else{
    rho_sum[((R+1)*z)+r] = rho_h[(0*(L+1)*(R+1))+((R+1)*z)+r] * (double) (r-1);

  }
}

__global__ void clear_courant(double *courant_array, int L, int R, int max_threads){
  int r = blockIdx.x%R;
  int z = (floorf(blockIdx.x/R) * max_threads) + threadIdx.x;
  
  if(r==0 || z==0 || r>=R || z>=L){
    return;
  }
  courant_array[((R+1)*z)+r]=0;
}

__global__ void print_sorted(double *courant_array, int L, int R){
  int count=0;
  for(int j=0; j<(L+1)*(R+1); j++){
    if(courant_array[j]>0){
      printf("%.5f\n",courant_array[j]);
      count++;
    }
  }
  printf("Number of points with non-zero courant number=%d\n", count);
}
extern "C" double get_signal_gpu(MJD_Siggen_Setup *setup, GPU_data *gpu_setup, int L, int R, int n_iter, double grid, int save_time, int num_threads){

  grid_sig=grid;
  passivated_surface_thickness_sig= setup->passivated_thickness;
  dz_correction=passivated_surface_thickness_sig/grid;

  int num_blocks = R * (ceil(L/num_threads)+1);
  double signal=0.f, hsum1=0.f, hsum2=0.f, esum1=0.f, esum2=0.f;


  cal_esum1<<<num_blocks, num_threads>>>(L, R, gpu_setup->rho_sum, gpu_setup->rho_e_gpu, gpu_setup->surface_rho_e, gpu_setup->wpot_gpu, num_threads);

  esum1 = thrust::reduce(thrust::device, gpu_setup->rho_sum, gpu_setup->rho_sum + (L+1)*(R+1));

  cal_esum2<<<num_blocks, num_threads>>>(L, R, gpu_setup->rho_sum, gpu_setup->rho_e_gpu, gpu_setup->surface_rho_e, num_threads);
  esum2 = thrust::reduce(thrust::device, gpu_setup->rho_sum, gpu_setup->rho_sum + (L+1)*(R+1));

  cal_hsum1<<<num_blocks, num_threads>>>(L, R, gpu_setup->rho_sum, gpu_setup->rho_h_gpu, gpu_setup->surface_rho_h, gpu_setup->wpot_gpu, num_threads);
  hsum1 = thrust::reduce(thrust::device, gpu_setup->rho_sum, gpu_setup->rho_sum + (L+1)*(R+1));


  cal_hsum2<<<num_blocks, num_threads>>>(L, R, gpu_setup->rho_sum, gpu_setup->rho_h_gpu, gpu_setup->surface_rho_h, num_threads);
  hsum2 = thrust::reduce(thrust::device, gpu_setup->rho_sum, gpu_setup->rho_sum + (L+1)*(R+1));
  
  if (n_iter > save_time && gpu_setup->hsum02 > hsum2) hsum1 += gpu_setup->hsum02 - hsum2;

  if (n_iter==save_time) {
      gpu_setup->esum01 = esum1; gpu_setup->esum02 = esum2;
      gpu_setup->hsum01 = hsum1; gpu_setup->hsum02 = hsum2;
    }


  //printf("hsum1=%.5f-hsum01=%.5f/hsum2=%.5f - esum1=%.5f-esum01=%.5f/esum2=%.5f,\n", hsum1, gpu_setup->hsum01, hsum2, esum1, gpu_setup->esum01, esum2);
  signal = 1000.0 * ((hsum1 - gpu_setup->hsum01) / gpu_setup->hsum02 - (esum1 - gpu_setup->esum01) / gpu_setup->esum02);
  //printf("Signals collected:%.5f\n", signal/1000);

  return signal;
}
  extern "C" double get_courant_number(GPU_data *gpu_setup, int L, int R, int num_blocks, int num_threads){

    // double courant_number;
    double courant_number = thrust::max_element(thrust::device_pointer_cast(gpu_setup->courant_array), thrust::device_pointer_cast(gpu_setup->courant_array) + (L+1)*(R+1))[0];
    // thrust::sort(thrust::device_pointer_cast(gpu_setup->courant_array), thrust::device_pointer_cast(gpu_setup->courant_array) + (L+1)*(R+1));
    // hipDeviceSynchronize();
    // print_sorted<<<1, 1>>>(gpu_setup->courant_array, L, R);
    // hipMemcpy(&courant_number, gpu_setup->courant_array+ (L+1)*(R+1)-1, sizeof(double), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    // printf("\nCourant number is %f\n", courant_number);

    // gpu_setup->courant_number_n=courant_number;
    clear_courant<<<num_blocks, num_threads>>>(gpu_setup->courant_array, L, R, num_threads);
    hipDeviceSynchronize();
    return courant_number;
  }