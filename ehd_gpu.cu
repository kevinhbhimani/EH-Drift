#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "mjd_siggen.h"
#include "detector_geometry.h"


/*
  Can be run as ./ehdrift config_files/P42575A.config -a 25.00 -z 0.10 -g P42575A -s 0.00
  WP can be calculated as ./ehdrift config_files/P42575A_calc_wp.config -a 15.00 -z 0.10 -g P42575A -s 0.00
  Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
  Max dimension size of a grid size (x,y,z): (65535, 65535, 65535)
*/

extern "C" int gpu_drift(MJD_Siggen_Setup *setup, int L, int R, float grid, float ***rho, float ***rho_test, int q, double *gone);
int drift_rho(MJD_Siggen_Setup *setup, int L, int R, float grid, float ***rho, int q, double *gone);

__managed__ float fq;
__managed__ int testVar;
__managed__ float drift_E[20]= {0.000,  100.,  160.,  240.,  300.,  500.,  600., 750.0, 1000., 1250., 1500., 1750., 2000., 2500., 3000., 3500., 4000., 4500., 5000., 1e10};
__managed__ int idid,idod,idd;
__managed__ double f_drift;

__managed__ float tstep;
__managed__ float delta;
__managed__ float delta_r;
__managed__ float wrap_around_radius;
__managed__ float ditch_thickness;
__managed__ float ditch_depth;
__managed__ float surface_drift_vel_factor;


__global__ void gpu_diffusion(int L, int R, float grid, float *rho,int q, double *v, char *point_type,  
  double *dr, double *dz, double *s1, double *s2, float *drift_offset, float *drift_slope, int max_threads, double *deltaez_array, double *deltaer_array){


  //printf("Time step is %d \n", setup-> step_time_calc);
  //formlaa for index n,z,r is (n*(L+1)*(R+1))+((R+1)*z)+r

  int r = blockIdx.x%R;
  int z = (floorf(blockIdx.x/R) * max_threads) + threadIdx.x;
  
  if(r==0 || z==0 || r>=R || z>=(L-2)){
    return;
  }
  int i, new_gpu_relax = 0;
  float E, E_r, E_z;
  double ve_z, ve_r, deltaez, deltaer;
  // if((r>=R) || (z>=L-2)){
  // printf("-----------Segfault error at radius=%d and z position=%d-----------\n",r,z);
  // }

  
  if (rho[(0*(L+1)*(R+1))+((R+1)*z)+r] < 1.0e-14) {
    rho[(1*(L+1)*(R+1))+((R+1)*z)+r] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r];
    return;
  }

  // calc E in r-direction
  if (r == 1) {  // r = 0; symmetry implies E_r = 0
    E_r = 0;
  } else if (point_type[((R+1)*z)+r] == CONTACT_EDGE) {
    E_r = ((v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1])*dr[(1*(L+1)*(R+1))+((R+1)*z)+r] +
          (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])*dr[(0*(L+1)*(R+1))+((R+1)*z)+r]) / (0.2*grid);
  } else if (point_type[((R+1)*z)+r] < INSIDE &&
            point_type[((R+1)*z)+r-1] == CONTACT_EDGE) {
    E_r =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r]) * dr[(1*(L+1)*(R+1))+((R+1)*z)+r-1] / ( 0.1*grid) ;
  } else if (point_type[((R+1)*z)+r] < INSIDE &&
            point_type[((R+1)*z)+r+1] == CONTACT_EDGE) {
    E_r =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1]) * dr[(0*(L+1)*(R+1))+((R+1)*z)+r+1] / ( 0.1*grid) ;
  } else if (r == R-1) {
    E_r = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])/(0.1*grid);
  } else {
    E_r = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1])/(0.2*grid);
  }
  // calc E in z-direction
  // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
  if (point_type[((R+1)*z)+r] == CONTACT_EDGE) {
    E_z = ((v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])*dz[(1*(L+1)*(R+1))+((R+1)*z)+r] +
          (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])*dz[(0*(L+1)*(R+1))+((R+1)*z)+r]) / (0.2*grid);
  } else if (point_type[((R+1)*z)+r] < INSIDE &&
            point_type[((R+1)*(z-1))+r] == CONTACT_EDGE) {
    E_z =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r]) * dz[(1*(L+1)*(R+1))+((R+1)*(z-1))+r] / ( 0.1*grid) ;
  } else if (point_type[((R+1)*z)+r] < INSIDE &&
            point_type[((R+1)*(z+1))+r] == CONTACT_EDGE) {
    E_z =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r]) * dz[(0*(L+1)*(R+1))+((R+1)*(z+1))+r] / ( 0.1*grid) ;
  } else if (z == 1) {
    E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])/(0.1*grid);
  } else if (z == L-1) {
    E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])/(0.1*grid);
  } else {
    E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])/(0.2*grid);
  }



  /* do diffusion to neighboring pixels */
  deltaez = deltaer = ve_z = ve_r = 0;

  if(z==6 && r==1253){
    printf("In GPU at 1, at z=%d and r=%d, the value of rho is %.7f deltaez is %.7f and deltaer is %.7f\n", z, r, rho[(1*(L+1)*(R+1))+((R+1)*z)+r], deltaez, deltaer);
  }

  E = fabs(E_z);
  if (E > 1.0) {
    for (i=0; E > drift_E[i+1]; i++);
    ve_z = (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
    deltaez = grid * ve_z * f_drift / E;
    }

  E = fabs(E_r);
  if (E > 1.0) {
    for (i=0; E > drift_E[i+1]; i++);
    ve_r = (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
    deltaer = grid * ve_r * f_drift / E;
    }

    if(z==6 && r==1253){
      printf("In GPU at 2, at z=%d and r=%d, the value of rho is %.7f deltaez is %.7f and deltaer is %.7f\n", z, r, rho[(1*(L+1)*(R+1))+((R+1)*z)+r], deltaez, deltaer);
    }
  if (0 && r == 100 && z == 10)
    printf("r z: %d %d; E_r deltaer: %f %f; E_z deltaez: %f %f; rho[0] = %f\n",
          r, z, E_r, deltaer, E_z, deltaez, rho[(0*(L+1)*(R+1))+((R+1)*z)+r]);

  /* reduce diffusion at passivated surfaces by a factor of surface_drift_vel_factor */
  if (1 &&
      ((r == idid && z < idd) ||
      (r < idid  && z == 1 ) ||
      (r >= idid && r <= idod && z == idd))) {
    // assume 2-micron-thick roughness/passivation in z
    deltaer *= surface_drift_vel_factor;
    deltaez *= surface_drift_vel_factor * grid/0.002; // * grid/0.002;
    }


  if (0 && z == 1) 
  printf("r,z = %d, %d E_r,z = %f, %f  deltaer,z = %f, %f  s1,s2 = %f, %f\n",
                    r, z, E_r, E_z, deltaer, deltaez, s1[r], s2[r]);

  if(z==6 && r==1253){
    printf("In GPU at 3, at z=%d and r=%d, the value of rho is %.7f deltaez is %.7f and deltaer is %.7f\n", z, r, rho[(1*(L+1)*(R+1))+((R+1)*z)+r], deltaez, deltaer);
  }

  deltaez_array[((R+1)*z)+r] = deltaez;
  deltaer_array[((R+1)*z)+r] = deltaer;

  // if (r < R-1 && point_type[((R+1)*z)+r+1] != DITCH) {
  //   rho[(1*(L+1)*(R+1))+((R+1)*z)+r+1] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s1[r] * (double) (r-1) / (double) (r);
  //   rho[(1*(L+1)*(R+1))+((R+1)*z)+r]   -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s1[r];
  //   //printf("value of rho at checkpoint 2 is %f \n",rho[(1*(L+1)*(R+1))+((R+1)*z)+r]);

  //   }
  // if (z > 1 && point_type[((R+1)*(z-1))+r] != DITCH) {
  //   rho[(1*(L+1)*(R+1))+((R+1)*(z-1))+r] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
  //   rho[(1*(L+1)*(R+1))+((R+1)*z)+r] -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
  //   }
  // if (z < L-1 && point_type[((R+1)*(z+1))+r] != DITCH) {
  //   rho[(1*(L+1)*(R+1))+((R+1)*(z+1))+r] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
  //   rho[(1*(L+1)*(R+1))+((R+1)*z)+r]   -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
  //   }
  // if (r > 2 && point_type[((R+1)*z)+r-1] != DITCH) {

  //   rho[(1*(L+1)*(R+1))+((R+1)*z)+(r-1)] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s2[r] * (double) (r-1) / (double) (r-2);
  //   rho[(1*(L+1)*(R+1))+((R+1)*z)+r]   -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s2[r];
  // }

}

  __global__ void diff_update(int L, int R, float *rho, char *point_type, double *s1, double *s2, double *deltaez_array, double *deltaer_array){
    for (int r=1; r<R; r++) {
      for (int z=1; z<L-2; z++) {
        if (rho[(0*(L+1)*(R+1))+((R+1)*z)+r] < 1.0e-14) {
          //printf("-----------EXCITING THE KERNAL-----------\n");
          continue;
        }
        double deltaez = deltaez_array[((R+1)*z)+r];
        double deltaer = deltaer_array[((R+1)*z)+r];

          // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
        rho[(1*(L+1)*(R+1))+((R+1)*z)+r] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r];

        if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
          printf("In GPU at 1, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[(1*(L+1)*(R+1))+((R+1)*z)+r]);
        }

        if (r < R-1 && point_type[((R+1)*z)+r+1] != DITCH) {
          rho[(1*(L+1)*(R+1))+((R+1)*z)+r+1] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s1[r] * (double) (r-1) / (double) (r);
          rho[(1*(L+1)*(R+1))+((R+1)*z)+r]   -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s1[r];
          //printf("value of rho at checkpoint 2 is %f \n",rho[(1*(L+1)*(R+1))+((R+1)*z)+r]);
      
          }

          if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
            printf("In GPU at 2, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[(1*(L+1)*(R+1))+((R+1)*z)+r]);
          }
        if (z > 1 && point_type[((R+1)*(z-1))+r] != DITCH) {
          rho[(1*(L+1)*(R+1))+((R+1)*(z-1))+r] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
          rho[(1*(L+1)*(R+1))+((R+1)*z)+r] -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
          }

          if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
            printf("In GPU at 3, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[(1*(L+1)*(R+1))+((R+1)*z)+r]);
          }
        if (z < L-1 && point_type[((R+1)*(z+1))+r] != DITCH) {
          rho[(1*(L+1)*(R+1))+((R+1)*(z+1))+r] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
          rho[(1*(L+1)*(R+1))+((R+1)*z)+r]   -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
          }

          if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
            printf("In GPU at 4, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[(1*(L+1)*(R+1))+((R+1)*z)+r]);
          }
        if (r > 2 && point_type[((R+1)*z)+r-1] != DITCH) {
      
          rho[(1*(L+1)*(R+1))+((R+1)*z)+(r-1)] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s2[r] * (double) (r-1) / (double) (r-2);
          rho[(1*(L+1)*(R+1))+((R+1)*z)+r]   -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s2[r];
        }

        if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
          printf("In GPU at 5, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[(1*(L+1)*(R+1))+((R+1)*z)+r]);
        }
      }
    }
    printf("In GPU at 7, at z=%d and r=%d, the value of rho is %.7f \n", 6, 1253, rho[(1*(L+1)*(R+1))+((R+1)*6)+1253]);

  }

  __global__ void gpu_self_repulsion(int L, int R, float grid, float *rho,int q, double *v, char *point_type,  double *dr, double *dz, 
    double *s1, double *s2, float *drift_offset, float *drift_slope, int max_threads, double *fr_array, double *fz_array, int *i_array, int *k_array){
  
      int new_gpu_relax = 0;
      float E, E_r, E_z;
      double dre, dze;
      double ve_z, ve_r;
  
      int r = blockIdx.x%R;
      int z = (floorf(blockIdx.x/R) * max_threads) + threadIdx.x;
      
      if(r==0 || z==0 || r>=R || z>=(L-2)){
        return;
      }
    
    if (rho[(1*(L+1)*(R+1))+((R+1)*z)+r] < 1.0e-14) {
      rho[(2*(L+1)*(R+1))+((R+1)*z)+r] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r];
      return;
    }
    // need to r-calculate all the fields
    // calc E in r-direction
    if (r == 1) {  // r = 0; symmetry implies E_r = 0
      E_r = 0;
    } else if (point_type[((R+1)*z)+r] == CONTACT_EDGE) {
      E_r = ((v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1])*dr[(1*(L+1)*(R+1))+((R+1)*z)+r] +
            (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])*dr[(0*(L+1)*(R+1))+((R+1)*z)+r]) / (0.2*grid);
    } else if (point_type[((R+1)*z)+r] < INSIDE &&
              point_type[((R+1)*z)+r-1] == CONTACT_EDGE) {
      E_r =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r]) * dr[(1*(L+1)*(R+1))+((R+1)*z)+r-1] / ( 0.1*grid) ;
    } else if (point_type[((R+1)*z)+r] < INSIDE &&
              point_type[((R+1)*z)+r+1] == CONTACT_EDGE) {
      E_r =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1]) * dr[(0*(L+1)*(R+1))+((R+1)*z)+r+1] / ( 0.1*grid) ;
    } else if (r == R-1) {
      E_r = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])/(0.1*grid);
    } else {
      E_r = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1])/(0.2*grid);
    }
    // calc E in z-direction
    // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
    if (point_type[((R+1)*z)+r] == CONTACT_EDGE) {
      E_z = ((v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])*dz[(1*(L+1)*(R+1))+((R+1)*z)+r] +
            (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])*dz[(0*(L+1)*(R+1))+((R+1)*z)+r]) / (0.2*grid);
    } else if (point_type[((R+1)*z)+r] < INSIDE &&
              point_type[((R+1)*(z-1))+r] == CONTACT_EDGE) {
      E_z =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r]) * dz[(1*(L+1)*(R+1))+((R+1)*(z-1))+r] / ( 0.1*grid) ;
    } else if (point_type[((R+1)*z)+r] < INSIDE &&
              point_type[((R+1)*(z+1))+r] == CONTACT_EDGE) {
      E_z =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r]) * dz[(0*(L+1)*(R+1))+((R+1)*(z+1))+r] / ( 0.1*grid) ;
    } else if (z == 1) {
      E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])/(0.1*grid);
    } else if (z == L-1) {
      E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])/(0.1*grid);
    } else {
      E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])/(0.2*grid);
    }
    ve_z = ve_r = 0;
    E = fabs(E_z);
    int b;
    if (E > 1.0) {
      for (b=0; E > drift_E[b+1]; b++);
      ve_z = fq * (drift_offset[b] + drift_slope[b]*(E - drift_E[b]));
    }
    E = fabs(E_r);
    if (E > 1.0) {
      for (b=0; E > drift_E[b+1]; b++);
      ve_r = fq * (drift_offset[b] + drift_slope[b]*(E - drift_E[b]));
    }
    /* reduce drift speed at passivated surfaces by a factor of surface_drift_vel_factor */
    if (1 &&
        ((r == idid && z < idd) ||
        (r < idid  && z == 1 ) ||
        (r >= idid && r <= idod && z == idd))) {
      ve_r *= surface_drift_vel_factor;
      ve_z *= surface_drift_vel_factor * grid/0.002;  // assume 2-micron-thick roughness/passivation in z
    }
  
  
    //-----------------------------------------------------------
  
    /* do drift to neighboring pixels */
    // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
    if (E_r > 0) {
      dre = -tstep*ve_r;
    } else {
      dre =  tstep*ve_r;
    }
    if (E_z > 0) {
      dze = -tstep*ve_z;
    } else {
      dze =  tstep*ve_z;
    }

    if (dre == 0.0) {
      i_array[((R+1)*z)+r]= r;
      fr_array[((R+1)*z)+r] = 1.0;
    } else {
      i_array[((R+1)*z)+r]= (double) r + dre;
      fr_array[((R+1)*z)+r] = ceil(dre) - dre;
    }
    if (i_array[((R+1)*z)+r]<1) {
      i_array[((R+1)*z)+r]= 1;
      fr_array[((R+1)*z)+r] = 1.0;
    }
    if (i_array[((R+1)*z)+r]>R-1) {
      i_array[((R+1)*z)+r]= R-1;
      fr_array[((R+1)*z)+r] = 0.0;
    }
    if (dre > 0 && z < idd && r <= idid && i_array[((R+1)*z)+r]>= idid) { // ditch ID
      i_array[((R+1)*z)+r]= idid;
      fr_array[((R+1)*z)+r] = 1.0;
    }
    if (dre < 0 && z < idd && r >= idod && i_array[((R+1)*z)+r]<= idod) { // ditch OD
      i_array[((R+1)*z)+r]= idod;
      fr_array[((R+1)*z)+r] = 0.0;
    }
  
    if (dze == 0.0) {
      k_array[((R+1)*z)+r]= z;
      fz_array[((R+1)*z)+r] = 1.0;
    } else {
      k_array[((R+1)*z)+r]= (double) z + dze;
      fz_array[((R+1)*z)+r] = ceil(dze) - dze;
    }
    if (k_array[((R+1)*z)+r]<1) {
      k_array[((R+1)*z)+r]= 1;
      fz_array[((R+1)*z)+r] = 1.0;
    }
    if (k_array[((R+1)*z)+r]>L-1) {
      k_array[((R+1)*z)+r]= L-1;
      fz_array[((R+1)*z)+r] = 0.0;
    }
    if (dze < 0 && r > idid && r < idod && k_array[((R+1)*z)+r]< idd) { // ditch depth
      k_array[((R+1)*z)+r]  = idd;
      fr_array[((R+1)*z)+r]  = 1.0;
    }
  
    
  
    if (1 && r == 100 && z == 10)
      printf("r z: %d %d; E_r i_array[((R+1)*z)+r]dre: %f %d %f; fr_array[((R+1)*z)+r] = %f\n"
            "r z: %d %d; E_z k_array[((R+1)*z)+r]dze: %f %d %f; fz_array[((R+1)*z)+r] = %f\n",
            r, z, E_r, i_array[((R+1)*z)+r], dre, fr_array[((R+1)*z)+r], r, z, E_z, k_array[((R+1)*z)+r], dze, fz_array[((R+1)*z)+r]);

  }
  
  __global__ void gpu_sr_update(int L, int R, float *rho, double *fr_array, double *fz_array, int *i_array, int *k_array){

    for (int r=1; r<R; r++) {
      for (int z=1; z<L-2; z++) {
      
      if (rho[(1*(L+1)*(R+1))+((R+1)*z)+r] < 1.0e-14) {
        continue;
      }

        if (i_array[((R+1)*z)+r]>=1 && i_array[((R+1)*z)+r]<R && k_array[((R+1)*z)+r]>=1 && k_array[((R+1)*z)+r]<L) {
          if (i_array[((R+1)*z)+r] > 1 && r > 1) {
            rho[(2*(L+1)*(R+1))+((R+1)*k_array[((R+1)*z)+r])+i_array[((R+1)*z)+r]] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr_array[((R+1)*z)+r]      *fz_array[((R+1)*z)+r]       * (double) (r-1) / (double) (i_array[((R+1)*z)+r]-1);
            rho[(2*(L+1)*(R+1))+((R+1)*k_array[((R+1)*z)+r])+i_array[((R+1)*z)+r]+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr_array[((R+1)*z)+r])*fz_array[((R+1)*z)+r]       * (double) (r-1) / (double) (i_array[((R+1)*z)+r]);
            rho[(2*(L+1)*(R+1))+((R+1)*(k_array[((R+1)*z)+r]+1))+i_array[((R+1)*z)+r]] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr_array[((R+1)*z)+r]      *(1.0-fz_array[((R+1)*z)+r]) * (double) (r-1) / (double) (i_array[((R+1)*z)+r]-1);
            rho[(2*(L+1)*(R+1))+((R+1)*(k_array[((R+1)*z)+r]+1))+i_array[((R+1)*z)+r]+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr_array[((R+1)*z)+r])*(1.0-fz_array[((R+1)*z)+r]) * (double) (r-1) / (double) (i_array[((R+1)*z)+r]);

          } 
          else if (i_array[((R+1)*z)+r] > 1) {  // r == 0
            rho[(2*(L+1)*(R+1))+((R+1)*k_array[((R+1)*z)+r])+i_array[((R+1)*z)+r]] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr_array[((R+1)*z)+r]      *fz_array[((R+1)*z)+r]       / (double) (8*i_array[((R+1)*z)+r]-8);
            rho[(2*(L+1)*(R+1))+((R+1)*k_array[((R+1)*z)+r])+i_array[((R+1)*z)+r]+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr_array[((R+1)*z)+r])*fz_array[((R+1)*z)+r]       / (double) (8*i_array[((R+1)*z)+r]);
            rho[(2*(L+1)*(R+1))+((R+1)*(k_array[((R+1)*z)+r]+1))+i_array[((R+1)*z)+r]] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr_array[((R+1)*z)+r]      *(1.0-fz_array[((R+1)*z)+r]) / (double) (8*i_array[((R+1)*z)+r]-8);
            rho[(2*(L+1)*(R+1))+((R+1)*(k_array[((R+1)*z)+r]+1))+i_array[((R+1)*z)+r]+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr_array[((R+1)*z)+r])*(1.0-fz_array[((R+1)*z)+r]) / (double) (8*i_array[((R+1)*z)+r]);
          } 
          else if (r > 1) {  // i_array[((R+1)*z)+r] == 0
            rho[(2*(L+1)*(R+1))+((R+1)*k_array[((R+1)*z)+r])+i_array[((R+1)*z)+r]] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr_array[((R+1)*z)+r]      *fz_array[((R+1)*z)+r]       * (double) (8*(R+1)-8);
            rho[(2*(L+1)*(R+1))+((R+1)*k_array[((R+1)*z)+r])+i_array[((R+1)*z)+r]+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr_array[((R+1)*z)+r])*fz_array[((R+1)*z)+r]       * (double) (r-1);
            rho[(2*(L+1)*(R+1))+((R+1)*(k_array[((R+1)*z)+r]+1))+i_array[((R+1)*z)+r]] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr_array[((R+1)*z)+r]      *(1.0-fz_array[((R+1)*z)+r]) * (double) (8*(R+1)-8);
            rho[(2*(L+1)*(R+1))+((R+1)*(k_array[((R+1)*z)+r]+1))+i_array[((R+1)*z)+r]+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr_array[((R+1)*z)+r])*(1.0-fz_array[((R+1)*z)+r]) * (double) (r-1);
          } 
          else {             // r == i_array[((R+1)*z)+r] == 0
            rho[(2*(L+1)*(R+1))+((R+1)*k_array[((R+1)*z)+r])+i_array[((R+1)*z)+r]] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr_array[((R+1)*z)+r]      *fz_array[((R+1)*z)+r];
            rho[(2*(L+1)*(R+1))+((R+1)*k_array[((R+1)*z)+r])+i_array[((R+1)*z)+r]+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr_array[((R+1)*z)+r])*fz_array[((R+1)*z)+r]       / 8.0; // vol_0 / vol_1 = 1/8
            rho[(2*(L+1)*(R+1))+((R+1)*(k_array[((R+1)*z)+r]+1))+i_array[((R+1)*z)+r]] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr_array[((R+1)*z)+r]      *(1.0-fz_array[((R+1)*z)+r]);
            rho[(2*(L+1)*(R+1))+((R+1)*(k_array[((R+1)*z)+r]+1))+i_array[((R+1)*z)+r]+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr_array[((R+1)*z)+r])*(1.0-fz_array[((R+1)*z)+r]) / 8.0;
          }
        }
      }
    }
  }

/* -------------------------------------- gpu_drift ------------------- */
// do the diffusion and drifting of the charge cloud densities
extern "C" int gpu_drift(MJD_Siggen_Setup *setup, int L, int R, float grid, float ***rho, float ***rho_test, int q, double *gone){
  
  tstep = setup-> step_time_calc;
  delta = 0.07*tstep;
  delta_r = 0.07*tstep;
  wrap_around_radius = setup->wrap_around_radius;
  ditch_thickness =  setup-> ditch_thickness;
  ditch_depth = setup-> ditch_depth;
  surface_drift_vel_factor = setup->surface_drift_vel_factor;

  grid = setup->xtal_grid;
    /* NOTE that impurity and field arrays in setup start at (i,j)=(1,1) for (r,z)=(0,0) */
    idid = lrint((wrap_around_radius - ditch_thickness)/grid) + 1; // ditch ID
    idod = lrint(wrap_around_radius/grid) + 1; // ditch OD
    idd =  lrint(ditch_depth/grid) + 1;        // ditch depth



   
    double *v_gpu;
    double *v_flat;
    v_flat = (double*)malloc(2*sizeof(double)*(L+1)*(R+1));
    hipMalloc((void**)&v_gpu, 2*sizeof(double)*(L+1)*(R+1));
    for(int i=0; i<2; i++) {
      for(int j=0; j<=L; j++){
        for(int k=0; k<=R; k++){
          v_flat[(i*(L+1)*(R+1))+((R+1)*j)+k] = setup->v[i][j][k];
        }
      }
    }
    hipMemcpy(v_gpu, v_flat, 2*sizeof(double)*(L+1)*(R+1), hipMemcpyHostToDevice);  


    char  *point_type_flat;
    char  *point_type_gpu;
    point_type_flat = (char*)malloc(sizeof(char)*(L+1)*(R+1));
    hipMalloc((void**)&point_type_gpu, sizeof(char)*(L+1)*(R+1));
    for(int j=0; j<=L; j++){
        for(int k=0; k<=R; k++){
          //printf("The value of point type at r = %d and z = %d is %.4c \n", k, j, setup->point_type[j][k]);
          point_type_flat[((R+1)*j)+k] = setup->point_type[j][k];
        }
      }
    hipMemcpy(point_type_gpu, point_type_flat, sizeof(char)*(L+1)*(R+1), hipMemcpyHostToDevice);
  
    double *dr_flat;
    double *dr_gpu;
    dr_flat = (double*)malloc(2*sizeof(double)*(L+1)*(R+1));
    hipMalloc((void**)&dr_gpu, 2*sizeof(double)*(L+1)*(R+1));
      for(int i=0; i<2; i++) {
        for(int j=1; j<=L; j++){
            for(int k=0; k<=R; k++){
              dr_flat[(i*(L+1)*(R+1))+((R+1)*j)+k] = setup->dr[i][j][k];
            }
          }
        }
    hipMemcpy(dr_gpu, dr_flat, 2*sizeof(double)*(L+1)*(R+1), hipMemcpyHostToDevice);
    
    double *dz_flat;
    double *dz_gpu;
    dz_flat = (double*)malloc(2*sizeof(double)*(L+1)*(R+1));
    hipMalloc((void**)&dz_gpu, 2*sizeof(double)*(L+1)*(R+1));
      for(int i=0; i<2; i++) {
        for(int j=1; j<=L; j++){
            for(int k=0; k<=R; k++){
              dz_flat[(i*(L+1)*(R+1))+((R+1)*j)+k] = setup->dz[i][j][k];
            }
          }
        }
    hipMemcpy(dz_gpu, dz_flat, 2*sizeof(double)*(L+1)*(R+1), hipMemcpyHostToDevice);
    
    double *s1_gpu, *s2_gpu;
    
    hipMalloc((void**)&s1_gpu, sizeof(double)*(R+1));
    hipMemcpy(s1_gpu, setup->s1, sizeof(double)*(R+1), hipMemcpyHostToDevice);
    
    
    hipMalloc((void**)&s2_gpu, sizeof(double)*(R+1));
    hipMemcpy(s2_gpu, setup->s2, sizeof(double)*(R+1), hipMemcpyHostToDevice);

    double *deltaez_array, *deltaer_array;
    hipMalloc((void**)&deltaez_array, sizeof(double)*(L+1)*(R+1));
    hipMalloc((void**)&deltaer_array, sizeof(double)*(L+1)*(R+1));

    double *fr_array, *fz_array;
    int *i_array, *k_array;
    hipMalloc((void**)&fr_array, sizeof(double)*(L+1)*(R+1));
    hipMalloc((void**)&fz_array, sizeof(double)*(L+1)*(R+1));
    hipMalloc((void**)&i_array, sizeof(int)*(L+1)*(R+1));
    hipMalloc((void**)&k_array, sizeof(int)*(L+1)*(R+1));


  // int index = 0;
  // for(int i=0; i<3; i++) {
  //   for(int j=0; j<L; j++){
  //     for(int k=0; k<R; k++){
  //       printf("Value at i=%d, j=%d, k=%d is=%f\n", i, j, k,rho[i][j][k]);
  //       printf("index is: %d\n",index);
  //       printf("Simulated index is: %d\n",(i*(L+1)*(R+1))+((R+1)*j)+k);
  //       index++;
  //     }
  //   }
  // }
 
    /* ASSUMPTIONS:
       0.1 mm grid size, 1 ns steps
       detector temperature = REF_TEMP = 77 K
       dealing only with electrons (no holes)
    */
    int i, r, z;

    float E_r, E_z;
    double ve_z, ve_r, deltaez, deltaer;
    
    fq = -q;
    deltaez = delta;
    deltaer = delta_r;


    float drift_offset_e[20]={0.0,   0.027, 0.038, 0.049 ,0.055, 0.074, 0.081,
          0.089, 0.101, 0.109, 0.116, 0.119, 0.122, 0.125,
          0.1275,0.1283,0.1288,0.1291,0.1293,0.1293};
    float drift_slope_e[20];

    float drift_offset_h[20]={0.0,   0.036, 0.047, 0.056, 0.06,  0.072, 0.077,
          0.081, 0.086, 0.089, 0.0925,0.095, 0.097, 0.1,
          0.1025,0.1036,0.1041,0.1045,0.1047,0.1047};
    float drift_slope_h[20];
    float *drift_offset, *drift_slope;
  
    for (i=0; i<20; i++) {
      drift_offset_e[i] /= grid;   // drift velocities in units of grid length
      drift_offset_h[i] /= grid;
    }
    for (i=0; i<19; i++) {
      drift_slope_e[i] = (drift_offset_e[i+1] - drift_offset_e[i]) /
                         (drift_E[i+1] - drift_E[i]);
      drift_slope_h[i] = (drift_offset_h[i+1] - drift_offset_h[i]) /
                         (drift_E[i+1] - drift_E[i]);
    }
    if (q < 0) { // electrons
      drift_offset = drift_offset_e;
      drift_slope  = drift_slope_e;
    } else {   // holes
      drift_offset = drift_offset_h;
      drift_slope  = drift_slope_h;
    }
  
    f_drift = 1.2e6; // * setup.xtal_temp/REF_TEMP;
    f_drift *= tstep / 4000.0;
    /* above is my own approximate parameterization of measurements of Jacoboni et al.
       1.2e6 * v_over_E   ~   D in cm2/s
       v_over_E = drift velocity / electric field   ~  mu
       note that Einstein's equation is D = mu*kT/e
       kT/e ~ 0.007/V ~ 0.07 mm/Vcm, => close enough to 0.12, okay
       For 20-micron bins and 1ns steps, DELTA = D / 4000
       For fixed D, DELTA goes as time_step_size/bin_size_squared
    */
    f_drift *= 0.02/grid * 0.02/grid; // correct for grid size
    // f *= 0.5;                   // artifically reduce diffusion to 50%
    E_r = E_z = 100; // just to get started; will change later
    for (i=0; E_z > drift_E[i+1]; i++);
    ve_z = fq * (drift_offset[i] + drift_slope[i]*(E_z - drift_E[i]))/E_z;
    deltaez = grid * ve_z * f_drift;
    for (i=0; E_r > drift_E[i+1]; i++);
    ve_r = fq * (drift_offset[i] + drift_slope[i]*(E_r - drift_E[i]))/E_r;
    deltaer = grid * ve_r * f_drift;
    printf ("D_z, D_r values (q=%d) at 100 V/cm: %f %f\n", q, deltaez, deltaer);
  

    for (z=0; z<L; z++) {
      for (r=0; r<R; r++) {
        rho_test[1][z][r] = rho_test[2][z][r] = 0;
      }
    }

  float *drift_offset_gpu, *drift_slope_gpu;
  
  hipMalloc((void**)&drift_offset_gpu, sizeof(float)*20);
  hipMemcpy(drift_offset_gpu, drift_offset, sizeof(float)*20, hipMemcpyHostToDevice);

  hipMalloc((void**)&drift_slope_gpu, sizeof(float)*20);
  hipMemcpy(drift_slope_gpu, drift_slope, sizeof(float)*20, hipMemcpyHostToDevice);

  float *rho_cpu_flat;
  float *rho_gpu;

  rho_cpu_flat = (float*)malloc(3*sizeof(float)*(L+1)*(R+1));
  hipMalloc((void**)&rho_gpu, 3*sizeof(float)*(L+1)*(R+1));

  for(int i=0; i<3; i++) {
    for(int j=0; j<L; j++){
        for(int k=0; k<R; k++){
          rho_cpu_flat[(i*(L+1)*(R+1))+((R+1)*j)+k]=rho_test[i][j][k];
        }
      }
    }


  // for(int i=0; i<3; i++) {
  //   //hipMemcpy(&rho_gpu[(i*(L+1)*(R+1))], &rho[i], sizeof(float)*(L+1), hipMemcpyHostToDevice);
  //   for(int j=0; j<L; j++){
  //     hipMemcpy(&rho_gpu[(i*(L+1)*(R+1))+((R+1)*j)], &rho[i][j], sizeof(float)*(R+1), hipMemcpyHostToDevice);
  //   }
  // }

  hipMemcpy(rho_gpu, rho_cpu_flat, 3*sizeof(float)*(L+1)*(R+1), hipMemcpyHostToDevice);


  // printf("Allocation and memory copy successfull\n");

  // printf("Executing the kernel\n");
  int num_threads = 300;
  int num_blocks = R * (ceil(L/num_threads)+1); //The +1 is just a precaution to make sure all R and Z values are included

  if(num_blocks<65535){
    gpu_diffusion<<<num_blocks,num_threads>>>(L, R, grid, rho_gpu, q, v_gpu, point_type_gpu, dr_gpu, dz_gpu, s1_gpu, s2_gpu, drift_offset_gpu, drift_slope_gpu, num_threads, deltaez_array, deltaer_array);
    hipDeviceSynchronize();
    diff_update<<<1,1>>>(L, R, rho_gpu, point_type_gpu, s1_gpu, s2_gpu, deltaez_array, deltaer_array);
    hipDeviceSynchronize();
    gpu_self_repulsion<<<num_blocks,num_threads>>>(L, R, grid, rho_gpu, q, v_gpu, point_type_gpu, dr_gpu, dz_gpu, s1_gpu, s2_gpu, drift_offset_gpu, drift_slope_gpu, num_threads, fr_array, fz_array, i_array, k_array);
    hipDeviceSynchronize();
    gpu_sr_update<<<1,1>>>(L, R, rho_gpu, fr_array, fz_array, i_array, k_array);
  }
  else{
    printf("----------------Pick a smaller block please----------------\n");
    return 0;
  }

  // printf("Done executing the kernel \n");

  hipDeviceSynchronize();


  // for(int o=0; o<3; o++) {
  //   //hipMemcpy(&rho[i], &rho_gpu[(i*(L+1)*(R+1))], sizeof(float)*(L+1), hipMemcpyDeviceToHost);
  //   for(int p=0; p<L; p++){
  //     hipMemcpy(&rho[o][p], &rho_gpu[(o*(L+1)*(R+1))+((R+1)*p)], sizeof(float)*(R+1), hipMemcpyDeviceToHost);
  //   }
  // }

      
  // printf("copying memory the rho density\n");
  hipMemcpy(rho_cpu_flat, rho_gpu, 3*sizeof(float)*(L+1)*(R+1), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  for(int i=0; i<3; i++) {
    for(int j=0; j<L; j++){
        for(int k=0; k<R; k++){
          memcpy(&rho_test[i][j][k], &rho_cpu_flat[(i*(L+1)*(R+1))+((R+1)*j)+k], sizeof(float));
        }
      }
    }


  for (z=0; z<L; z++) {
    for (r=0; r<R; r++) {
      if (setup->point_type[z][r] <= HVC) {
        //*gone += rho_test[2][z][r] * r;
        rho_test[2][z][r] = 0;
      }
    }
  }


  drift_rho(setup, L, R, grid, rho, q, gone);

  // printf("R=%d, Z=%d\n",R,L);
  printf("\n--------Running tests to compare the outcomes of CPU and GPU--------\n");

  #define MAX_ERR 1e-7
  int error_count = 0;
  for(int i=0; i<3; i++) {
    for(int j=0; j<L; j++){
      for(int k=0; k<R; k++){
        if(fabs(rho[i][j][k] - rho_test[i][j][k]) > MAX_ERR){
          error_count++;
          printf("Error at i=%d, z=%d and r=%d, actual answer is %f, calculated answer is %f \n",i,j,k, rho[i][j][k], rho_test[i][j][k]);
        }
      }
    }
  }
  if(error_count>0){
    printf("Total number of errors = %d \n", error_count);
  }
  else{
    printf("No errors found!!!!\n\n");
  }

  // for(int i=0; i<3; i++) {
  //   for(int j=0; j<L; j++){
  //       free(rho_test[i][j]);
  //     }
  //   }


  hipFree(rho_gpu);
  hipFree(v_gpu);
  hipFree(point_type_gpu);
  hipFree(dr_gpu);
  hipFree(dz_gpu);
  hipFree(s1_gpu);
  hipFree(s2_gpu);
  hipFree(fr_array);
  hipFree(fz_array);
  hipFree(i_array);
  hipFree(k_array);
  hipFree(deltaez_array);
  hipFree(deltaer_array);

  hipFree(drift_offset_gpu);
  hipFree(drift_slope_gpu);
  free(rho_cpu_flat);
  free(v_flat);
  free(point_type_flat);
  free(dr_flat);
  free(dz_flat);

  return 0;

}

/* -------------------------------------- drift_rho ------------------- */
// do the diffusion and drifting of the charge cloud densities

int drift_rho(MJD_Siggen_Setup *setup, int L, int R, float grid, float ***rho,
  int q, double *gone) {

  #define TSTEP   setup-> step_time_calc  // time step of calculation, in ns; do not exceed 2.0 !!!
  #define DELTA   (0.07*TSTEP) /* Prob. of a charge moving to next 20-micron bin during a time step */
  #define DELTA_R (0.07*TSTEP) /* ... in r-direction */

  /* ASSUMPTIONS:
  0.1 mm grid size, 1 ns steps
  detector temperature = REF_TEMP = 77 K
  dealing only with electrons (no holes)
  */

  int    i, k, r, z, new_var = 0;
  double dre, dze;
  double ***v = setup->v;
  float  E_r, E_z, E, fq = -q;
  double ve_z, ve_r, f, fr, fz, deltaez = DELTA, deltaer = DELTA_R;
  float drift_E[20]=       {0.000,  100.,  160.,  240.,  300.,  500.,  600.,
  750.0, 1000., 1250., 1500., 1750., 2000., 2500.,
  3000., 3500., 4000., 4500., 5000., 1e10};
  float drift_offset_e[20]={0.0,   0.027, 0.038, 0.049 ,0.055, 0.074, 0.081,
  0.089, 0.101, 0.109, 0.116, 0.119, 0.122, 0.125,
  0.1275,0.1283,0.1288,0.1291,0.1293,0.1293};
  float drift_slope_e[20];

  float drift_offset_h[20]={0.0,   0.036, 0.047, 0.056, 0.06,  0.072, 0.077,
  0.081, 0.086, 0.089, 0.0925,0.095, 0.097, 0.1,
  0.1025,0.1036,0.1041,0.1045,0.1047,0.1047};
  float drift_slope_h[20];
  float *drift_offset, *drift_slope;


  for (i=0; i<20; i++) {
  drift_offset_e[i] /= grid;   // drift velocities in units of grid length
  drift_offset_h[i] /= grid;
  }
  for (i=0; i<19; i++) {
  drift_slope_e[i] = (drift_offset_e[i+1] - drift_offset_e[i]) /
            (drift_E[i+1] - drift_E[i]);
  drift_slope_h[i] = (drift_offset_h[i+1] - drift_offset_h[i]) /
            (drift_E[i+1] - drift_E[i]);
  }
  if (q < 0) { // electrons
  drift_offset = drift_offset_e;
  drift_slope  = drift_slope_e;
  } else {   // holes
  drift_offset = drift_offset_h;
  drift_slope  = drift_slope_h;
  }

  f = 1.2e6; // * setup.xtal_temp/REF_TEMP;
  f *= TSTEP / 4000.0;
  /* above is my own approximate parameterization of measurements of Jacoboni et al.
  1.2e6 * v_over_E   ~   D in cm2/s
  v_over_E = drift velocity / electric field   ~  mu
  note that Einstein's equation is D = mu*kT/e
  kT/e ~ 0.007/V ~ 0.07 mm/Vcm, => close enough to 0.12, okay
  For 20-micron bins and 1ns steps, DELTA = D / 4000
  For fixed D, DELTA goes as time_step_size/bin_size_squared
  */
  f *= 0.02/grid * 0.02/grid; // correct for grid size
  // f *= 0.5;                   // artifically reduce diffusion to 50%

  E_r = E_z = 100; // just to get started; will change later
  for (i=0; E_z > drift_E[i+1]; i++);
  ve_z = fq * (drift_offset[i] + drift_slope[i]*(E_z - drift_E[i]))/E_z;
  deltaez = grid * ve_z * f;
  for (i=0; E_r > drift_E[i+1]; i++);
  ve_r = fq * (drift_offset[i] + drift_slope[i]*(E_r - drift_E[i]))/E_r;
  deltaer = grid * ve_r * f;
  printf ("D_z, D_r values (q=%d) at 100 V/cm: %f %f\n", q, deltaez, deltaer);

  for (z=0; z<L; z++) {
  for (r=0; r<R; r++) {
  rho[1][z][r] = rho[2][z][r] = 0;
  }
  }
  /* NOTE that impurity and field arrays in setup start at (i,j)=(1,1) for (r,z)=(0,0) */
  int idid = lrint((setup->wrap_around_radius - setup->ditch_thickness)/grid) + 1; // ditch ID
  int idod = lrint(setup->wrap_around_radius/grid) + 1; // ditch OD
  int idd =  lrint(setup->ditch_depth/grid) + 1;        // ditch depth
  for (r=1; r<R; r++) {
    for (z=1; z<L-2; z++) {
      if (rho[0][z][r] < 1.0e-14) {
        rho[1][z][r] += rho[0][z][r];
        continue;
      }
      // calc E in r-direction
      if (r == 1) {  // r = 0; symmetry implies E_r = 0
        E_r = 0;
      } 
      else if (setup->point_type[z][r] == CONTACT_EDGE) {
        E_r = ((v[new_var][z][r] - v[new_var][z][r+1])*setup->dr[1][z][r] +
        (v[new_var][z][r-1] - v[new_var][z][r])*setup->dr[0][z][r]) / (0.2*grid);
      } 
      else if (setup->point_type[z][r] < INSIDE &&
          setup->point_type[z][r-1] == CONTACT_EDGE) {
        E_r =  (v[new_var][z][r-1] - v[new_var][z][r]) * setup->dr[1][z][r-1] / ( 0.1*grid) ;
      } 
      else if (setup->point_type[z][r] < INSIDE &&
          setup->point_type[z][r+1] == CONTACT_EDGE) {
        E_r =  (v[new_var][z][r] - v[new_var][z][r+1]) * setup->dr[0][z][r+1] / ( 0.1*grid) ;
      } 
      else if (r == R-1) {
        E_r = (v[new_var][z][r-1] - v[new_var][z][r])/(0.1*grid);
      } 
      else {
        E_r = (v[new_var][z][r-1] - v[new_var][z][r+1])/(0.2*grid);
      }
      // calc E in z-direction
      // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
      if (setup->point_type[z][r] == CONTACT_EDGE) {
        E_z = ((v[new_var][z][r] - v[new_var][z+1][r])*setup->dz[1][z][r] +
        (v[new_var][z-1][r] - v[new_var][z][r])*setup->dz[0][z][r]) / (0.2*grid);
      } 
      else if (setup->point_type[z][r] < INSIDE &&
          setup->point_type[z-1][r] == CONTACT_EDGE) {
        E_z =  (v[new_var][z-1][r] - v[new_var][z][r]) * setup->dz[1][z-1][r] / ( 0.1*grid) ;
      } 
      else if (setup->point_type[z][r] < INSIDE &&
          setup->point_type[z+1][r] == CONTACT_EDGE) {
        E_z =  (v[new_var][z][r] - v[new_var][z+1][r]) * setup->dz[0][z+1][r] / ( 0.1*grid) ;
      } 
      else if (z == 1) {
        E_z = (v[new_var][z][r] - v[new_var][z+1][r])/(0.1*grid);
      } 
      else if (z == L-1) {
        E_z = (v[new_var][z-1][r] - v[new_var][z][r])/(0.1*grid);
      } 
      else {
        E_z = (v[new_var][z-1][r] - v[new_var][z+1][r])/(0.2*grid);
      }

      /* do diffusion to neighboring pixels */
      deltaez = deltaer = ve_z = ve_r = 0;

      // if(z==6 && r==1253){
      //   printf("In CPU at 1, at z=%d and r=%d, the value of rho is %.7f deltaez is %.7f and deltaer is %.7f\n", z, r, rho[1][z][r], deltaez, deltaer);
      // }

      E = fabs(E_z);
      if (E > 1.0) {
      for (i=0; E > drift_E[i+1]; i++);
      ve_z = (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
      deltaez = grid * ve_z * f / E;
      }

      E = fabs(E_r);
      if (E > 1.0) {
      for (i=0; E > drift_E[i+1]; i++);
      ve_r = (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
      deltaer = grid * ve_r * f / E;
      }
      if (0 && r == 100 && z == 10)
      printf("r z: %d %d; E_r deltaer: %f %f; E_z deltaez: %f %f; rho[0] = %f\n",
        r, z, E_r, deltaer, E_z, deltaez, rho[0][z][r]);

      /* reduce diffusion at passivated surfaces by a factor of surface_drift_vel_factor */
      if (1 &&
      ((r == idid && z < idd) ||
      (r < idid  && z == 1 ) ||
      (r >= idid && r <= idod && z == idd))) {
      // assume 2-micron-thick roughness/passivation in z
      deltaer *= setup->surface_drift_vel_factor;
      deltaez *= setup->surface_drift_vel_factor * grid/0.002; // * grid/0.002;
      }


      if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
        printf("In CPU at 1, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[1][z][r]);
      }
      // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
      rho[1][z][r]   += rho[0][z][r];
      if (0 && z == 1) printf("r,z = %d, %d E_r,z = %f, %f  deltaer,z = %f, %f  s1,s2 = %f, %f\n",
                  r, z, E_r, E_z, deltaer, deltaez, setup->s1[r], setup->s2[r]);

                  if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
                    printf("In CPU at 2, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[1][z][r]);
      }

      if (r < R-1 && setup->point_type[z][r+1] != DITCH) {
      //if (setup->point_type[z][r+1] > HVC)
      rho[1][z][r+1] += rho[0][z][r]*deltaer * setup->s1[r] * (double) (r-1) / (double) (r);
      rho[1][z][r]   -= rho[0][z][r]*deltaer * setup->s1[r];
      }

      if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
        printf("In CPU at 3, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[1][z][r]);
      }

      if (z > 1 && setup->point_type[z-1][r] != DITCH) {
      //if (setup->point_type[z-1][r] > HVC)
      rho[1][z-1][r] += rho[0][z][r]*deltaez;
      rho[1][z][r]   -= rho[0][z][r]*deltaez;
      }

      if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
        printf("In CPU at 4, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[1][z][r]);
      }
      if (z < L-1 && setup->point_type[z+1][r] != DITCH) {
      //if (setup->point_type[z+1][r] > HVC)
      rho[1][z+1][r] += rho[0][z][r]*deltaez;
      rho[1][z][r]   -= rho[0][z][r]*deltaez;
      }

      if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
        printf("In CPU at 5, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[1][z][r]);
      }
      if (r > 2 && setup->point_type[z][r-1] != DITCH) {
      //if (setup->point_type[z][r-1] > HVC)
      rho[1][z][r-1] += rho[0][z][r]*deltaer * setup->s2[r] * (double) (r-1) / (double) (r-2);
      rho[1][z][r]   -= rho[0][z][r]*deltaer * setup->s2[r];
      }

      if((z==6||z==5||z==4) && (r==1253||r==1252||r==1254)){
        printf("In CPU at 6, at z=%d and r=%d, the value of rho is %.7f \n", z, r, rho[1][z][r]);
      }

      if(rho[1][6][1253]-14.731561<0.000001){
        printf("The change happnds at z=%d and r=%d \n", z, r);
      }

      //-----------------------------------------------------------
    }
  }

  printf("In CPU at 7, at z=%d and r=%d, the value of rho is %.7f \n", 6, 1253, rho[1][6][1253]);

    

  for (r=1; r<R; r++) {
    for (z=1; z<L-2; z++) {
      if (rho[1][z][r] < 1.0e-14) {
        rho[2][z][r] += rho[1][z][r];
        continue;
      }
      // need to r-calculate all the fields
      // calc E in r-direction
      if (r == 1) {  // r = 0; symmetry implies E_r = 0
      E_r = 0;
      } 
      else if (setup->point_type[z][r] == CONTACT_EDGE) {
      E_r = ((v[new_var][z][r] - v[new_var][z][r+1])*setup->dr[1][z][r] +
        (v[new_var][z][r-1] - v[new_var][z][r])*setup->dr[0][z][r]) / (0.2*grid);
      } 
      else if (setup->point_type[z][r] < INSIDE &&
          setup->point_type[z][r-1] == CONTACT_EDGE) {
      E_r =  (v[new_var][z][r-1] - v[new_var][z][r]) * setup->dr[1][z][r-1] / ( 0.1*grid) ;
      } 
      else if (setup->point_type[z][r] < INSIDE &&
          setup->point_type[z][r+1] == CONTACT_EDGE) {
      E_r =  (v[new_var][z][r] - v[new_var][z][r+1]) * setup->dr[0][z][r+1] / ( 0.1*grid) ;
      } 
      else if (r == R-1) {
      E_r = (v[new_var][z][r-1] - v[new_var][z][r])/(0.1*grid);
      } 
      else {
      E_r = (v[new_var][z][r-1] - v[new_var][z][r+1])/(0.2*grid);
      }
      // calc E in z-direction
      // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
      if (setup->point_type[z][r] == CONTACT_EDGE) {
      E_z = ((v[new_var][z][r] - v[new_var][z+1][r])*setup->dz[1][z][r] +
        (v[new_var][z-1][r] - v[new_var][z][r])*setup->dz[0][z][r]) / (0.2*grid);
      } 
      else if (setup->point_type[z][r] < INSIDE &&
          setup->point_type[z-1][r] == CONTACT_EDGE) {
      E_z =  (v[new_var][z-1][r] - v[new_var][z][r]) * setup->dz[1][z-1][r] / ( 0.1*grid) ;
      } 
      else if (setup->point_type[z][r] < INSIDE &&
          setup->point_type[z+1][r] == CONTACT_EDGE) {
      E_z =  (v[new_var][z][r] - v[new_var][z+1][r]) * setup->dz[0][z+1][r] / ( 0.1*grid) ;
      } 
      else if (z == 1) {
      E_z = (v[new_var][z][r] - v[new_var][z+1][r])/(0.1*grid);
      } 
      else if (z == L-1) {
      E_z = (v[new_var][z-1][r] - v[new_var][z][r])/(0.1*grid);
      } 
      else {
      E_z = (v[new_var][z-1][r] - v[new_var][z+1][r])/(0.2*grid);
      }
      ve_z = ve_r = 0;
      E = fabs(E_z);
      if (E > 1.0) {
      for (i=0; E > drift_E[i+1]; i++);
      ve_z = fq * (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
      }
      E = fabs(E_r);
      if (E > 1.0) {
      for (i=0; E > drift_E[i+1]; i++);
      ve_r = fq * (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
      }
      /* reduce drift speed at passivated surfaces by a factor of surface_drift_vel_factor */
      if (1 &&
      ((r == idid && z < idd) ||
      (r < idid  && z == 1 ) ||
      (r >= idid && r <= idod && z == idd))) {
      ve_r *= setup->surface_drift_vel_factor;
      ve_z *= setup->surface_drift_vel_factor * grid/0.002;  // assume 2-micron-thick roughness/passivation in z
      }


      //-----------------------------------------------------------

      /* do drift to neighboring pixels */
      // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
      if (E_r > 0) {
      dre = -TSTEP*ve_r;
      } 
      else {
      dre =  TSTEP*ve_r;
      }
      if (E_z > 0) {
      dze = -TSTEP*ve_z;
      } 
      else {
      dze =  TSTEP*ve_z;
      }

      if (dre == 0.0) {
      i = r;
      fr = 1.0;
      } 
      else {
      i = (double) r + dre;
      fr = ceil(dre) - dre;
      }
      if (i<1) {
      i = 1;
      fr = 1.0;
      }
      if (i>R-1) {
      i = R-1;
      fr = 0.0;
      }
      if (dre > 0 && z < idd && r <= idid && i >= idid) { // ditch ID
      i = idid;
      fr = 1.0;
      }
      if (dre < 0 && z < idd && r >= idod && i <= idod) { // ditch OD
      i = idod;
      fr = 0.0;
      }

      if (dze == 0.0) {
      k = z;
      fz = 1.0;
      } 
      else {
      k = (double) z + dze;
      fz = ceil(dze) - dze;
      }
      if (k<1) {
      k = 1;
      fz = 1.0;
      }
      if (k>L-1) {
      k = L-1;
      fz = 0.0;
      }
      if (dze < 0 && r > idid && r < idod && k < idd) { // ditch depth
      k   = idd;
      fr  = 1.0;
      }
      if (0 && r == 100 && z == 10)
      printf("r z: %d %d; E_r i dre: %f %d %f; fr = %f\n"
        "r z: %d %d; E_z k dze: %f %d %f; fz = %f\n",
        r, z, E_r, i, dre, fr, r, z, E_z, k, dze, fz);

      if (i>=1 && i<R && k>=1 && k<L) {
        if (i > 1 && r > 1) {
          rho[2][k  ][i  ] += rho[1][z][r] * fr      *fz       * (double) (r-1) / (double) (i-1);
          rho[2][k  ][i+1] += rho[1][z][r] * (1.0-fr)*fz       * (double) (r-1) / (double) (i);
          rho[2][k+1][i  ] += rho[1][z][r] * fr      *(1.0-fz) * (double) (r-1) / (double) (i-1);
          rho[2][k+1][i+1] += rho[1][z][r] * (1.0-fr)*(1.0-fz) * (double) (r-1) / (double) (i);
        } 
        else if (i > 1) {  // r == 0
          rho[2][k  ][i  ] += rho[1][z][r] * fr      *fz       / (double) (8*i-8);
          rho[2][k  ][i+1] += rho[1][z][r] * (1.0-fr)*fz       / (double) (8*i);
          rho[2][k+1][i  ] += rho[1][z][r] * fr      *(1.0-fz) / (double) (8*i-8);
          rho[2][k+1][i+1] += rho[1][z][r] * (1.0-fr)*(1.0-fz) / (double) (8*i);
        } 
        else if (r > 1) {  // i == 0
          rho[2][k  ][i  ] += rho[1][z][r] * fr      *fz       * (double) (8*(R+1)-8);
          rho[2][k  ][i+1] += rho[1][z][r] * (1.0-fr)*fz       * (double) (r-1);
          rho[2][k+1][i  ] += rho[1][z][r] * fr      *(1.0-fz) * (double) (8*(R+1)-8);
          rho[2][k+1][i+1] += rho[1][z][r] * (1.0-fr)*(1.0-fz) * (double) (r-1);
        } 
        else {             // r == i == 0
          rho[2][k  ][i  ] += rho[1][z][r] * fr      *fz;
          rho[2][k  ][i+1] += rho[1][z][r] * (1.0-fr)*fz       / 8.0; // vol_0 / vol_1 = 1/8
          rho[2][k+1][i  ] += rho[1][z][r] * fr      *(1.0-fz);
          rho[2][k+1][i+1] += rho[1][z][r] * (1.0-fr)*(1.0-fz) / 8.0;
        }
      }
    }
  }

  for (z=0; z<L; z++) {
    for (r=0; r<R; r++) {
      if (setup->point_type[z][r] <= HVC) {
        *gone += rho[2][z][r] * r;
        rho[2][z][r] = 0;
      }
  }
}


return 0;
}