#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "mjd_siggen.h"
#include "detector_geometry.h"


/*
Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
Max dimension size of a grid size (x,y,z): (65535, 65535, 65535)
*/

extern "C" int gpu_drift(MJD_Siggen_Setup *setup, int L, int R, float grid, float ***rho, float ***rho_test, int q, double *gone);
int drift_rho(MJD_Siggen_Setup *setup, int L, int R, float grid, float ***rho, int q, double *gone);

__managed__ float fq;
__managed__ int testVar;
__managed__ float drift_E[20]= {0.000,  100.,  160.,  240.,  300.,  500.,  600., 750.0, 1000., 1250., 1500., 1750., 2000., 2500., 3000., 3500., 4000., 4500., 5000., 1e10};
__managed__ int idid,idod,idd;
__managed__ double f_drift;

__managed__ float tstep;
__managed__ float delta;
__managed__ float delta_r;
__managed__ float wrap_around_radius;
__managed__ float ditch_thickness;
__managed__ float ditch_depth;
__managed__ float surface_drift_vel_factor;


__global__ void gpu_diffusion(int L, int R, float grid, float *rho,int q, double *v, char *point_type,  
  double *dr, double *dz, double *s1, double *s2, float *drift_offset, float *drift_slope, int max_threads){


  //printf("Time step is %d \n", setup-> step_time_calc);
  //formlaa for index n,z,r is (n*(L+1)*(R+1))+((R+1)*z)+r

  int r = blockIdx.x%R;
  int z = (floorf(blockIdx.x/R) * max_threads) + threadIdx.x;
  
  if(r==0 || z==0 || r>=R || z>=(L-2)){
    return;
  }
  int i, new_gpu_relax = 0;
  float E, E_r, E_z;
  double ve_z, ve_r, deltaez, deltaer;
  // if((r>=R) || (z>=L-2)){
  // printf("-----------Segfault error at radius=%d and z position=%d-----------\n",r,z);
  // }

  //printf("I am in radius=%d and z position=%d\n",r,z);
  
  if (rho[(0*(L+1)*(R+1))+((R+1)*z)+r] < 1.0e-14) {
    rho[(1*(L+1)*(R+1))+((R+1)*z)+r] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r];
    //printf("-----------EXCITING THE KERNAL-----------\n");
    return;
  }
  // calc E in r-direction
  if (r == 1) {  // r = 0; symmetry implies E_r = 0
    E_r = 0;
  } else if (point_type[((R+1)*z)+r] == CONTACT_EDGE) {
    E_r = ((v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1])*dr[(1*(L+1)*(R+1))+((R+1)*z)+r] +
          (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])*dr[(0*(L+1)*(R+1))+((R+1)*z)+r]) / (0.2*grid);
  } else if (point_type[((R+1)*z)+r] < INSIDE &&
            point_type[((R+1)*z)+r-1] == CONTACT_EDGE) {
    E_r =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r]) * dr[(1*(L+1)*(R+1))+((R+1)*z)+r-1] / ( 0.1*grid) ;
  } else if (point_type[((R+1)*z)+r] < INSIDE &&
            point_type[((R+1)*z)+r+1] == CONTACT_EDGE) {
    E_r =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1]) * dr[(0*(L+1)*(R+1))+((R+1)*z)+r+1] / ( 0.1*grid) ;
  } else if (r == R-1) {
    E_r = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])/(0.1*grid);
  } else {
    E_r = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1])/(0.2*grid);
  }
  // calc E in z-direction
  // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
  if (point_type[((R+1)*z)+r] == CONTACT_EDGE) {
    E_z = ((v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])*dz[(1*(L+1)*(R+1))+((R+1)*z)+r] +
          (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])*dz[(0*(L+1)*(R+1))+((R+1)*z)+r]) / (0.2*grid);
  } else if (point_type[((R+1)*z)+r] < INSIDE &&
            point_type[((R+1)*(z-1))+r] == CONTACT_EDGE) {
    E_z =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r]) * dz[(1*(L+1)*(R+1))+((R+1)*(z-1))+r] / ( 0.1*grid) ;
  } else if (point_type[((R+1)*z)+r] < INSIDE &&
            point_type[((R+1)*(z+1))+r] == CONTACT_EDGE) {
    E_z =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r]) * dz[(0*(L+1)*(R+1))+((R+1)*(z+1))+r] / ( 0.1*grid) ;
  } else if (z == 1) {
    E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])/(0.1*grid);
  } else if (z == L-1) {
    E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])/(0.1*grid);
  } else {
    E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])/(0.2*grid);
  }



  /* do diffusion to neighboring pixels */
  deltaez = deltaer = ve_z = ve_r = 0;
  E = fabs(E_z);
  if (E > 1.0) {
    for (i=0; E > drift_E[i+1]; i++);
    ve_z = (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
    deltaez = grid * ve_z * f_drift / E;
    }
  E = fabs(E_r);
  if (E > 1.0) {
    for (i=0; E > drift_E[i+1]; i++);
    ve_r = (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
    deltaer = grid * ve_r * f_drift / E;
    }
  if (0 && r == 100 && z == 10)
    printf("r z: %d %d; E_r deltaer: %f %f; E_z deltaez: %f %f; rho[0] = %f\n",
          r, z, E_r, deltaer, E_z, deltaez, rho[(0*(L+1)*(R+1))+((R+1)*z)+r]);

  /* reduce diffusion at passivated surfaces by a factor of surface_drift_vel_factor */
  if (0 &&
      ((r == idid && z < idd) ||
      (r < idid  && z == 1 ) ||
      (r >= idid && r <= idod && z == idd))) {
    // assume 2-micron-thick roughness/passivation in z
    deltaer *= surface_drift_vel_factor;
    deltaez *= surface_drift_vel_factor * grid/0.002; // * grid/0.002;
    }

  // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
  rho[(1*(L+1)*(R+1))+((R+1)*z)+r] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r];
  if (0 && z == 1) 
  printf("r,z = %d, %d E_r,z = %f, %f  deltaer,z = %f, %f  s1,s2 = %f, %f\n",
                    r, z, E_r, E_z, deltaer, deltaez, s1[r], s2[r]);

  
  if (r < R-1 && point_type[((R+1)*z)+r+1] != DITCH) {
    rho[(1*(L+1)*(R+1))+((R+1)*z)+r+1] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s1[r] * (double) (r-1) / (double) (r);
    rho[(1*(L+1)*(R+1))+((R+1)*z)+r]   -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s1[r];
    //printf("value of rho at checkpoint 2 is %f \n",rho[(1*(L+1)*(R+1))+((R+1)*z)+r]);

    }
  if (z > 1 && point_type[((R+1)*(z-1))+r] != DITCH) {
    rho[(1*(L+1)*(R+1))+((R+1)*(z-1))+r] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
    rho[(1*(L+1)*(R+1))+((R+1)*z)+r] -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
    }
  if (z < L-1 && point_type[((R+1)*(z+1))+r] != DITCH) {
    rho[(1*(L+1)*(R+1))+((R+1)*(z+1))+r] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
    rho[(1*(L+1)*(R+1))+((R+1)*z)+r]   -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaez;
    }
  if (r > 2 && point_type[((R+1)*z)+r-1] != DITCH) {

    rho[(1*(L+1)*(R+1))+((R+1)*z)+(r-1)] += rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s2[r] * (double) (r-1) / (double) (r-2);
    rho[(1*(L+1)*(R+1))+((R+1)*z)+r]   -= rho[(0*(L+1)*(R+1))+((R+1)*z)+r]*deltaer * s2[r];
    }

  }




  __global__ void gpu_self_repulsion(int L, int R, float grid, float *rho,int q, double *v, char *point_type,  
    double *dr, double *dz, double *s1, double *s2, float *drift_offset, float *drift_slope, int max_threads){
  
      int i,k, new_gpu_relax = 0;
      float E, E_r, E_z;
      double dre, dze, fr, fz;
      double ve_z, ve_r;
  
      int r = blockIdx.x%R;
      int z = (floorf(blockIdx.x/R) * max_threads) + threadIdx.x;
      
      if(r==0 || z==0 || r>=R || z>=(L-2)){
        return;
      }
    
    if (rho[(1*(L+1)*(R+1))+((R+1)*z)+r] < 1.0e-14) {
      rho[(2*(L+1)*(R+1))+((R+1)*z)+r] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r];
      return;
    }
    // need to r-calculate all the fields
    // calc E in r-direction
    if (r == 1) {  // r = 0; symmetry implies E_r = 0
      E_r = 0;
    } else if (point_type[((R+1)*z)+r] == CONTACT_EDGE) {
      E_r = ((v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1])*dr[(1*(L+1)*(R+1))+((R+1)*z)+r] +
            (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])*dr[(0*(L+1)*(R+1))+((R+1)*z)+r]) / (0.2*grid);
    } else if (point_type[((R+1)*z)+r] < INSIDE &&
              point_type[((R+1)*z)+r-1] == CONTACT_EDGE) {
      E_r =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r]) * dr[(1*(L+1)*(R+1))+((R+1)*z)+r-1] / ( 0.1*grid) ;
    } else if (point_type[((R+1)*z)+r] < INSIDE &&
              point_type[((R+1)*z)+r+1] == CONTACT_EDGE) {
      E_r =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1]) * dr[(0*(L+1)*(R+1))+((R+1)*z)+r+1] / ( 0.1*grid) ;
    } else if (r == R-1) {
      E_r = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])/(0.1*grid);
    } else {
      E_r = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r-1] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r+1])/(0.2*grid);
    }
    // calc E in z-direction
    // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
    if (point_type[((R+1)*z)+r] == CONTACT_EDGE) {
      E_z = ((v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])*dz[(1*(L+1)*(R+1))+((R+1)*z)+r] +
            (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])*dz[(0*(L+1)*(R+1))+((R+1)*z)+r]) / (0.2*grid);
    } else if (point_type[((R+1)*z)+r] < INSIDE &&
              point_type[((R+1)*(z-1))+r] == CONTACT_EDGE) {
      E_z =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r]) * dz[(1*(L+1)*(R+1))+((R+1)*(z-1))+r] / ( 0.1*grid) ;
    } else if (point_type[((R+1)*z)+r] < INSIDE &&
              point_type[((R+1)*(z+1))+r] == CONTACT_EDGE) {
      E_z =  (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r]) * dz[(0*(L+1)*(R+1))+((R+1)*(z+1))+r] / ( 0.1*grid) ;
    } else if (z == 1) {
      E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])/(0.1*grid);
    } else if (z == L-1) {
      E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*z)+r])/(0.1*grid);
    } else {
      E_z = (v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z-1))+r] - v[(new_gpu_relax*(L+1)*(R+1))+((R+1)*(z+1))+r])/(0.2*grid);
    }
    ve_z = ve_r = 0;
    E = fabs(E_z);
    if (E > 1.0) {
      for (i=0; E > drift_E[i+1]; i++);
      ve_z = fq * (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
    }
    E = fabs(E_r);
    if (E > 1.0) {
      for (i=0; E > drift_E[i+1]; i++);
      ve_r = fq * (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
    }
    /* reduce drift speed at passivated surfaces by a factor of surface_drift_vel_factor */
    if (1 &&
        ((r == idid && z < idd) ||
        (r < idid  && z == 1 ) ||
        (r >= idid && r <= idod && z == idd))) {
      ve_r *= surface_drift_vel_factor;
      ve_z *= surface_drift_vel_factor * grid/0.002;  // assume 2-micron-thick roughness/passivation in z
    }
  
  
    //-----------------------------------------------------------
  
    /* do drift to neighboring pixels */
    // enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
    if (E_r > 0) {
      dre = -tstep*ve_r;
    } else {
      dre =  tstep*ve_r;
    }
    if (E_z > 0) {
      dze = -tstep*ve_z;
    } else {
      dze =  tstep*ve_z;
    }
  
    if (dre == 0.0) {
      i = r;
      fr = 1.0;
    } else {
      i = (double) r + dre;
      fr = ceil(dre) - dre;
    }
    if (i<1) {
      i = 1;
      fr = 1.0;
    }
    if (i>R-1) {
      i = R-1;
      fr = 0.0;
    }
    if (dre > 0 && z < idd && r <= idid && i >= idid) { // ditch ID
      i = idid;
      fr = 1.0;
    }
    if (dre < 0 && z < idd && r >= idod && i <= idod) { // ditch OD
      i = idod;
      fr = 0.0;
    }
  
    if (dze == 0.0) {
      k = z;
      fz = 1.0;
    } else {
      k = (double) z + dze;
      fz = ceil(dze) - dze;
    }
    if (k<1) {
      k = 1;
      fz = 1.0;
    }
    if (k>L-1) {
      k = L-1;
      fz = 0.0;
    }
    if (dze < 0 && r > idid && r < idod && k < idd) { // ditch depth
      k   = idd;
      fr  = 1.0;
    }
  
    
  
    if (1 && r == 100 && z == 10)
      printf("r z: %d %d; E_r i dre: %f %d %f; fr = %f\n"
            "r z: %d %d; E_z k dze: %f %d %f; fz = %f\n",
            r, z, E_r, i, dre, fr, r, z, E_z, k, dze, fz);


    // if(z==2 && r==501){
    //   printf("CP 1 in GPU rho[2][%d][%d] is %f\n", z, r, rho[(2*(L+1)*(R+1))+((R+1)*z)+r]);
    //   printf("Value of i=%d and k=%d\n",i,k);
    // }


    // if (i>=1 && i<R && k>=1 && k<L) {
    //   if (i > 1 && r > 1) {

    //     // if(z==2 && r==501){
    //     //   printf("CP 1 in GPU rho[2][%d][%d] is %f\n", z, r, rho[(2*(L+1)*(R+1))+((R+1)*z)+r]);
    //     // }
    //     rho[(2*(L+1)*(R+1))+((R+1)*k)+i] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr      *fz       * (double) (r-1) / (double) (i-1);
    //     rho[(2*(L+1)*(R+1))+((R+1)*k)+i+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr)*fz       * (double) (r-1) / (double) (i);
    //     rho[(2*(L+1)*(R+1))+((R+1)*(k+1))+i] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr      *(1.0-fz) * (double) (r-1) / (double) (i-1);
    //     rho[(2*(L+1)*(R+1))+((R+1)*(k+1))+i+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr)*(1.0-fz) * (double) (r-1) / (double) (i);

    //     // if(z==2 && r==501){
    //     //   printf("CP 1 in GPU rho[2][%d][%d] is %f\n", z, r, rho[(2*(L+1)*(R+1))+((R+1)*z)+r]);
    //     // }
    //   } else if (i > 1) {  // r == 0
    //     rho[(2*(L+1)*(R+1))+((R+1)*k)+i] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr      *fz       / (double) (8*i-8);
    //     rho[(2*(L+1)*(R+1))+((R+1)*k)+i+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr)*fz       / (double) (8*i);
    //     rho[(2*(L+1)*(R+1))+((R+1)*(k+1))+i] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr      *(1.0-fz) / (double) (8*i-8);
    //     rho[(2*(L+1)*(R+1))+((R+1)*(k+1))+i+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr)*(1.0-fz) / (double) (8*i);
    //   } else if (r > 1) {  // i == 0
    //     rho[(2*(L+1)*(R+1))+((R+1)*k)+i] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr      *fz       * (double) (8*(R+1)-8);
    //     rho[(2*(L+1)*(R+1))+((R+1)*k)+i+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr)*fz       * (double) (r-1);
    //     rho[(2*(L+1)*(R+1))+((R+1)*(k+1))+i] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr      *(1.0-fz) * (double) (8*(R+1)-8);
    //     rho[(2*(L+1)*(R+1))+((R+1)*(k+1))+i+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr)*(1.0-fz) * (double) (r-1);
    //   } else {             // r == i == 0
    //     rho[(2*(L+1)*(R+1))+((R+1)*k)+i] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr      *fz;
    //     rho[(2*(L+1)*(R+1))+((R+1)*k)+i+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr)*fz       / 8.0; // vol_0 / vol_1 = 1/8
    //     rho[(2*(L+1)*(R+1))+((R+1)*(k+1))+i] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * fr      *(1.0-fz);
    //     rho[(2*(L+1)*(R+1))+((R+1)*(k+1))+i+1] += rho[(1*(L+1)*(R+1))+((R+1)*z)+r] * (1.0-fr)*(1.0-fz) / 8.0;
    //   }
    // }


  }

/* -------------------------------------- gpu_drift ------------------- */
// do the diffusion and drifting of the charge cloud densities
extern "C" int gpu_drift(MJD_Siggen_Setup *setup, int L, int R, float grid, float ***rho, float ***rho_test, int q, double *gone){
  
  tstep = setup-> step_time_calc;
  delta = 0.07*tstep;
  delta_r = 0.07*tstep;
  wrap_around_radius = setup->wrap_around_radius;
  ditch_thickness =  setup-> ditch_thickness;
  ditch_depth = setup-> ditch_depth;
  surface_drift_vel_factor = setup->surface_drift_vel_factor;

  grid = setup->xtal_grid;
    /* NOTE that impurity and field arrays in setup start at (i,j)=(1,1) for (r,z)=(0,0) */
    idid = lrint((wrap_around_radius - ditch_thickness)/grid) + 1; // ditch ID
    idod = lrint(wrap_around_radius/grid) + 1; // ditch OD
    idd =  lrint(ditch_depth/grid) + 1;        // ditch depth



   
    double *v_gpu;
    double *v_flat;
    v_flat = (double*)malloc(2*sizeof(double)*(L+1)*(R+1));
    hipMalloc((void**)&v_gpu, 2*sizeof(double)*(L+1)*(R+1));
    for(int i=0; i<2; i++) {
      for(int j=0; j<=L; j++){
        for(int k=0; k<=R; k++){
          v_flat[(i*(L+1)*(R+1))+((R+1)*j)+k] = setup->v[i][j][k];
        }
      }
    }
    hipMemcpy(v_gpu, v_flat, 2*sizeof(double)*(L+1)*(R+1), hipMemcpyHostToDevice);  


    char  *point_type_flat;
    char  *point_type_gpu;
    point_type_flat = (char*)malloc(sizeof(char)*(L+1)*(R+1));
    hipMalloc((void**)&point_type_gpu, sizeof(char)*(L+1)*(R+1));
    for(int j=0; j<=L; j++){
        for(int k=0; k<=R; k++){
          //printf("The value of point type at r = %d and z = %d is %.4c \n", k, j, setup->point_type[j][k]);
          point_type_flat[((R+1)*j)+k] = setup->point_type[j][k];
        }
      }
    hipMemcpy(point_type_gpu, point_type_flat, sizeof(char)*(L+1)*(R+1), hipMemcpyHostToDevice);
  
    double *dr_flat;
    double *dr_gpu;
    dr_flat = (double*)malloc(2*sizeof(double)*(L+1)*(R+1));
    hipMalloc((void**)&dr_gpu, 2*sizeof(double)*(L+1)*(R+1));
      for(int i=0; i<2; i++) {
        for(int j=1; j<=L; j++){
            for(int k=0; k<=R; k++){
              dr_flat[(i*(L+1)*(R+1))+((R+1)*j)+k] = setup->dr[i][j][k];
            }
          }
        }
    hipMemcpy(dr_gpu, dr_flat, 2*sizeof(double)*(L+1)*(R+1), hipMemcpyHostToDevice);
    
    double *dz_flat;
    double *dz_gpu;
    dz_flat = (double*)malloc(2*sizeof(double)*(L+1)*(R+1));
    hipMalloc((void**)&dz_gpu, 2*sizeof(double)*(L+1)*(R+1));
      for(int i=0; i<2; i++) {
        for(int j=1; j<=L; j++){
            for(int k=0; k<=R; k++){
              dz_flat[(i*(L+1)*(R+1))+((R+1)*j)+k] = setup->dz[i][j][k];
            }
          }
        }
    hipMemcpy(dz_gpu, dz_flat, 2*sizeof(double)*(L+1)*(R+1), hipMemcpyHostToDevice);
    
    double *s1_gpu, *s2_gpu;
    
    hipMalloc((void**)&s1_gpu, sizeof(double)*(R+1));
    hipMemcpy(s1_gpu, setup->s1, sizeof(double)*(R+1), hipMemcpyHostToDevice);
    
    
    hipMalloc((void**)&s2_gpu, sizeof(double)*(R+1));
    hipMemcpy(s2_gpu, setup->s2, sizeof(double)*(R+1), hipMemcpyHostToDevice);

  // int index = 0;
  // for(int i=0; i<3; i++) {
  //   for(int j=0; j<L; j++){
  //     for(int k=0; k<R; k++){
  //       printf("Value at i=%d, j=%d, k=%d is=%f\n", i, j, k,rho[i][j][k]);
  //       printf("index is: %d\n",index);
  //       printf("Simulated index is: %d\n",(i*(L+1)*(R+1))+((R+1)*j)+k);
  //       index++;
  //     }
  //   }
  // }
 
    /* ASSUMPTIONS:
       0.1 mm grid size, 1 ns steps
       detector temperature = REF_TEMP = 77 K
       dealing only with electrons (no holes)
    */
    int i, r, z;

    float E_r, E_z;
    double ve_z, ve_r, deltaez, deltaer;
    
    fq = -q;
    deltaez = delta;
    deltaer = delta_r;


    float drift_offset_e[20]={0.0,   0.027, 0.038, 0.049 ,0.055, 0.074, 0.081,
          0.089, 0.101, 0.109, 0.116, 0.119, 0.122, 0.125,
          0.1275,0.1283,0.1288,0.1291,0.1293,0.1293};
    float drift_slope_e[20];

    float drift_offset_h[20]={0.0,   0.036, 0.047, 0.056, 0.06,  0.072, 0.077,
          0.081, 0.086, 0.089, 0.0925,0.095, 0.097, 0.1,
          0.1025,0.1036,0.1041,0.1045,0.1047,0.1047};
    float drift_slope_h[20];
    float *drift_offset, *drift_slope;
  
    for (i=0; i<20; i++) {
      drift_offset_e[i] /= grid;   // drift velocities in units of grid length
      drift_offset_h[i] /= grid;
    }
    for (i=0; i<19; i++) {
      drift_slope_e[i] = (drift_offset_e[i+1] - drift_offset_e[i]) /
                         (drift_E[i+1] - drift_E[i]);
      drift_slope_h[i] = (drift_offset_h[i+1] - drift_offset_h[i]) /
                         (drift_E[i+1] - drift_E[i]);
    }
    if (q < 0) { // electrons
      drift_offset = drift_offset_e;
      drift_slope  = drift_slope_e;
    } else {   // holes
      drift_offset = drift_offset_h;
      drift_slope  = drift_slope_h;
    }
  
    f_drift = 1.2e6; // * setup.xtal_temp/REF_TEMP;
    f_drift *= tstep / 4000.0;
    /* above is my own approximate parameterization of measurements of Jacoboni et al.
       1.2e6 * v_over_E   ~   D in cm2/s
       v_over_E = drift velocity / electric field   ~  mu
       note that Einstein's equation is D = mu*kT/e
       kT/e ~ 0.007/V ~ 0.07 mm/Vcm, => close enough to 0.12, okay
       For 20-micron bins and 1ns steps, DELTA = D / 4000
       For fixed D, DELTA goes as time_step_size/bin_size_squared
    */
    f_drift *= 0.02/grid * 0.02/grid; // correct for grid size
    // f *= 0.5;                   // artifically reduce diffusion to 50%
    E_r = E_z = 100; // just to get started; will change later
    for (i=0; E_z > drift_E[i+1]; i++);
    ve_z = fq * (drift_offset[i] + drift_slope[i]*(E_z - drift_E[i]))/E_z;
    deltaez = grid * ve_z * f_drift;
    for (i=0; E_r > drift_E[i+1]; i++);
    ve_r = fq * (drift_offset[i] + drift_slope[i]*(E_r - drift_E[i]))/E_r;
    deltaer = grid * ve_r * f_drift;
    printf ("D_z, D_r values (q=%d) at 100 V/cm: %f %f\n", q, deltaez, deltaer);
  

    for (z=0; z<L; z++) {
      for (r=0; r<R; r++) {
        rho_test[1][z][r] = rho_test[2][z][r] = 0;
      }
    }

  float *drift_offset_gpu, *drift_slope_gpu;
  
  hipMalloc((void**)&drift_offset_gpu, sizeof(float)*20);
  hipMemcpy(drift_offset_gpu, drift_offset, sizeof(float)*20, hipMemcpyHostToDevice);

  hipMalloc((void**)&drift_slope_gpu, sizeof(float)*20);
  hipMemcpy(drift_slope_gpu, drift_slope, sizeof(float)*20, hipMemcpyHostToDevice);

  float *rho_cpu_flat;
  float *rho_gpu;

  rho_cpu_flat = (float*)malloc(3*sizeof(float)*(L+1)*(R+1));
  hipMalloc((void**)&rho_gpu, 3*sizeof(float)*(L+1)*(R+1));

  for(int i=0; i<3; i++) {
    for(int j=0; j<L; j++){
        for(int k=0; k<R; k++){
          rho_cpu_flat[(i*(L+1)*(R+1))+((R+1)*j)+k]=rho_test[i][j][k];
        }
      }
    }


  // for(int i=0; i<3; i++) {
  //   //hipMemcpy(&rho_gpu[(i*(L+1)*(R+1))], &rho[i], sizeof(float)*(L+1), hipMemcpyHostToDevice);
  //   for(int j=0; j<L; j++){
  //     hipMemcpy(&rho_gpu[(i*(L+1)*(R+1))+((R+1)*j)], &rho[i][j], sizeof(float)*(R+1), hipMemcpyHostToDevice);
  //   }
  // }

  hipMemcpy(rho_gpu, rho_cpu_flat, 3*sizeof(float)*(L+1)*(R+1), hipMemcpyHostToDevice);


  // printf("Allocation and memory copy successfull\n");

  // printf("Executing the kernel\n");
  int num_threads = 300;
  int num_blocks = R * (ceil(L/num_threads)+1); //The +1 is just a precaution to make sure all R and Z values are included

  if(num_blocks<65535){
    gpu_diffusion<<<num_blocks,num_threads>>>(L, R, grid, rho_gpu, q, v_gpu, point_type_gpu, dr_gpu, dz_gpu, s1_gpu, s2_gpu, drift_offset_gpu, drift_slope_gpu, num_threads);
    hipDeviceSynchronize();
    gpu_self_repulsion<<<num_blocks,num_threads>>>(L, R, grid, rho_gpu, q, v_gpu, point_type_gpu, dr_gpu, dz_gpu, s1_gpu, s2_gpu, drift_offset_gpu, drift_slope_gpu, num_threads);
  }
  else{
    printf("----------------Pick a smaller block please----------------\n");
    return 0;
  }

  // printf("Done executing the kernel \n");

  hipDeviceSynchronize();


  // for(int o=0; o<3; o++) {
  //   //hipMemcpy(&rho[i], &rho_gpu[(i*(L+1)*(R+1))], sizeof(float)*(L+1), hipMemcpyDeviceToHost);
  //   for(int p=0; p<L; p++){
  //     hipMemcpy(&rho[o][p], &rho_gpu[(o*(L+1)*(R+1))+((R+1)*p)], sizeof(float)*(R+1), hipMemcpyDeviceToHost);
  //   }
  // }

      
  // printf("copying memory the rho density\n");
  hipMemcpy(rho_cpu_flat, rho_gpu, 3*sizeof(float)*(L+1)*(R+1), hipMemcpyDeviceToHost);

  for(int i=0; i<3; i++) {
    for(int j=0; j<L; j++){
        for(int k=0; k<R; k++){
          memcpy(&rho_test[i][j][k], &rho_cpu_flat[(i*(L+1)*(R+1))+((R+1)*j)+k], sizeof(float));
        }
      }
    }

  for (z=0; z<L; z++) {
    for (r=0; r<R; r++) {
      if (setup->point_type[z][r] <= HVC) {
        //*gone += rho_test[2][z][r] * r;
        rho_test[2][z][r] = 0;
      }
    }
  }


  drift_rho(setup, L, R, grid, rho, q, gone);

  printf("R=%d, Z=%d\n",R,L);
  printf("\n--------Running tests to compare the outcomes of CPU and GPU--------\n");

  #define MAX_ERR 1e-6
  int error_count = 0;
  for(int i=0; i<3; i++) {
    for(int j=0; j<L; j++){
      for(int k=0; k<R; k++){
        if(fabs(rho[i][j][k] - rho_test[i][j][k]) > MAX_ERR){
          error_count++;
          printf("Error at i=%d, z=%d and r=%d, actual answer is %f, calculated answer is %f \n",i,j,k, rho[i][j][k], rho_test[i][j][k]);
        }
      }
    }
  }
  if(error_count>0){
    printf("Total number of errors = %d \n", error_count);
  }
  else{
    printf("No errors found!!!!\n\n");
  }

  // for(int i=0; i<3; i++) {
  //   for(int j=0; j<L; j++){
  //       free(rho_test[i][j]);
  //     }
  //   }


  hipFree(rho_gpu);
  hipFree(v_gpu);
  hipFree(point_type_gpu);
  hipFree(dr_gpu);
  hipFree(dz_gpu);
  hipFree(s1_gpu);
  hipFree(s2_gpu);
  hipFree(drift_offset_gpu);
  hipFree(drift_slope_gpu);
  free(rho_cpu_flat);
  free(v_flat);
  free(point_type_flat);
  free(dr_flat);
  free(dz_flat);

  return 0;

}

/* -------------------------------------- drift_rho ------------------- */
// do the diffusion and drifting of the charge cloud densities

int drift_rho(MJD_Siggen_Setup *setup, int L, int R, float grid, float ***rho,
  int q, double *gone) {

#define TSTEP   setup-> step_time_calc  // time step of calculation, in ns; do not exceed 2.0 !!!
#define DELTA   (0.07*TSTEP) /* Prob. of a charge moving to next 20-micron bin during a time step */
#define DELTA_R (0.07*TSTEP) /* ... in r-direction */

/* ASSUMPTIONS:
0.1 mm grid size, 1 ns steps
detector temperature = REF_TEMP = 77 K
dealing only with electrons (no holes)
*/

int    i, k, r, z, new_var = 0;
double dre, dze;
double ***v = setup->v;
float  E_r, E_z, E, fq = -q;
double ve_z, ve_r, f, fr, fz, deltaez = DELTA, deltaer = DELTA_R;
float drift_E[20]=       {0.000,  100.,  160.,  240.,  300.,  500.,  600.,
750.0, 1000., 1250., 1500., 1750., 2000., 2500.,
3000., 3500., 4000., 4500., 5000., 1e10};
float drift_offset_e[20]={0.0,   0.027, 0.038, 0.049 ,0.055, 0.074, 0.081,
0.089, 0.101, 0.109, 0.116, 0.119, 0.122, 0.125,
0.1275,0.1283,0.1288,0.1291,0.1293,0.1293};
float drift_slope_e[20];

float drift_offset_h[20]={0.0,   0.036, 0.047, 0.056, 0.06,  0.072, 0.077,
0.081, 0.086, 0.089, 0.0925,0.095, 0.097, 0.1,
0.1025,0.1036,0.1041,0.1045,0.1047,0.1047};
float drift_slope_h[20];
float *drift_offset, *drift_slope;


for (i=0; i<20; i++) {
drift_offset_e[i] /= grid;   // drift velocities in units of grid length
drift_offset_h[i] /= grid;
}
for (i=0; i<19; i++) {
drift_slope_e[i] = (drift_offset_e[i+1] - drift_offset_e[i]) /
           (drift_E[i+1] - drift_E[i]);
drift_slope_h[i] = (drift_offset_h[i+1] - drift_offset_h[i]) /
           (drift_E[i+1] - drift_E[i]);
}
if (q < 0) { // electrons
drift_offset = drift_offset_e;
drift_slope  = drift_slope_e;
} else {   // holes
drift_offset = drift_offset_h;
drift_slope  = drift_slope_h;
}

f = 1.2e6; // * setup.xtal_temp/REF_TEMP;
f *= TSTEP / 4000.0;
/* above is my own approximate parameterization of measurements of Jacoboni et al.
1.2e6 * v_over_E   ~   D in cm2/s
v_over_E = drift velocity / electric field   ~  mu
note that Einstein's equation is D = mu*kT/e
kT/e ~ 0.007/V ~ 0.07 mm/Vcm, => close enough to 0.12, okay
For 20-micron bins and 1ns steps, DELTA = D / 4000
For fixed D, DELTA goes as time_step_size/bin_size_squared
*/
f *= 0.02/grid * 0.02/grid; // correct for grid size
// f *= 0.5;                   // artifically reduce diffusion to 50%

E_r = E_z = 100; // just to get started; will change later
for (i=0; E_z > drift_E[i+1]; i++);
ve_z = fq * (drift_offset[i] + drift_slope[i]*(E_z - drift_E[i]))/E_z;
deltaez = grid * ve_z * f;
for (i=0; E_r > drift_E[i+1]; i++);
ve_r = fq * (drift_offset[i] + drift_slope[i]*(E_r - drift_E[i]))/E_r;
deltaer = grid * ve_r * f;
printf ("D_z, D_r values (q=%d) at 100 V/cm: %f %f\n", q, deltaez, deltaer);

for (z=0; z<L; z++) {
for (r=0; r<R; r++) {
rho[1][z][r] = rho[2][z][r] = 0;
}
}
/* NOTE that impurity and field arrays in setup start at (i,j)=(1,1) for (r,z)=(0,0) */
int idid = lrint((setup->wrap_around_radius - setup->ditch_thickness)/grid) + 1; // ditch ID
int idod = lrint(setup->wrap_around_radius/grid) + 1; // ditch OD
int idd =  lrint(setup->ditch_depth/grid) + 1;        // ditch depth
for (r=1; r<R; r++) {
for (z=1; z<L-2; z++) {
if (rho[0][z][r] < 1.0e-14) {
rho[1][z][r] += rho[0][z][r];
continue;
}
// calc E in r-direction
if (r == 1) {  // r = 0; symmetry implies E_r = 0
E_r = 0;
} 
else if (setup->point_type[z][r] == CONTACT_EDGE) {
E_r = ((v[new_var][z][r] - v[new_var][z][r+1])*setup->dr[1][z][r] +
   (v[new_var][z][r-1] - v[new_var][z][r])*setup->dr[0][z][r]) / (0.2*grid);
} 
else if (setup->point_type[z][r] < INSIDE &&
     setup->point_type[z][r-1] == CONTACT_EDGE) {
E_r =  (v[new_var][z][r-1] - v[new_var][z][r]) * setup->dr[1][z][r-1] / ( 0.1*grid) ;
} 
else if (setup->point_type[z][r] < INSIDE &&
     setup->point_type[z][r+1] == CONTACT_EDGE) {
E_r =  (v[new_var][z][r] - v[new_var][z][r+1]) * setup->dr[0][z][r+1] / ( 0.1*grid) ;
} 
else if (r == R-1) {
E_r = (v[new_var][z][r-1] - v[new_var][z][r])/(0.1*grid);
} 
else {
E_r = (v[new_var][z][r-1] - v[new_var][z][r+1])/(0.2*grid);
}
// calc E in z-direction
// enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
if (setup->point_type[z][r] == CONTACT_EDGE) {
E_z = ((v[new_var][z][r] - v[new_var][z+1][r])*setup->dz[1][z][r] +
   (v[new_var][z-1][r] - v[new_var][z][r])*setup->dz[0][z][r]) / (0.2*grid);
} 
else if (setup->point_type[z][r] < INSIDE &&
     setup->point_type[z-1][r] == CONTACT_EDGE) {
E_z =  (v[new_var][z-1][r] - v[new_var][z][r]) * setup->dz[1][z-1][r] / ( 0.1*grid) ;
} 
else if (setup->point_type[z][r] < INSIDE &&
     setup->point_type[z+1][r] == CONTACT_EDGE) {
E_z =  (v[new_var][z][r] - v[new_var][z+1][r]) * setup->dz[0][z+1][r] / ( 0.1*grid) ;
} 
else if (z == 1) {
E_z = (v[new_var][z][r] - v[new_var][z+1][r])/(0.1*grid);
} 
else if (z == L-1) {
E_z = (v[new_var][z-1][r] - v[new_var][z][r])/(0.1*grid);
} 
else {
E_z = (v[new_var][z-1][r] - v[new_var][z+1][r])/(0.2*grid);
}

/* do diffusion to neighboring pixels */
deltaez = deltaer = ve_z = ve_r = 0;
E = fabs(E_z);
if (E > 1.0) {
for (i=0; E > drift_E[i+1]; i++);
ve_z = (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
deltaez = grid * ve_z * f / E;
}
E = fabs(E_r);
if (E > 1.0) {
for (i=0; E > drift_E[i+1]; i++);
ve_r = (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
deltaer = grid * ve_r * f / E;
}
if (0 && r == 100 && z == 10)
printf("r z: %d %d; E_r deltaer: %f %f; E_z deltaez: %f %f; rho[0] = %f\n",
   r, z, E_r, deltaer, E_z, deltaez, rho[0][z][r]);

/* reduce diffusion at passivated surfaces by a factor of surface_drift_vel_factor */
if (1 &&
((r == idid && z < idd) ||
(r < idid  && z == 1 ) ||
(r >= idid && r <= idod && z == idd))) {
// assume 2-micron-thick roughness/passivation in z
deltaer *= setup->surface_drift_vel_factor;
deltaez *= setup->surface_drift_vel_factor * grid/0.002; // * grid/0.002;
}

// enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
rho[1][z][r]   += rho[0][z][r];
if (0 && z == 1) printf("r,z = %d, %d E_r,z = %f, %f  deltaer,z = %f, %f  s1,s2 = %f, %f\n",
             r, z, E_r, E_z, deltaer, deltaez, setup->s1[r], setup->s2[r]);
if (r < R-1 && setup->point_type[z][r+1] != DITCH) {
//if (setup->point_type[z][r+1] > HVC)
rho[1][z][r+1] += rho[0][z][r]*deltaer * setup->s1[r] * (double) (r-1) / (double) (r);
rho[1][z][r]   -= rho[0][z][r]*deltaer * setup->s1[r];
}
if (z > 1 && setup->point_type[z-1][r] != DITCH) {
//if (setup->point_type[z-1][r] > HVC)
rho[1][z-1][r] += rho[0][z][r]*deltaez;
rho[1][z][r]   -= rho[0][z][r]*deltaez;
}
if (z < L-1 && setup->point_type[z+1][r] != DITCH) {
//if (setup->point_type[z+1][r] > HVC)
rho[1][z+1][r] += rho[0][z][r]*deltaez;
rho[1][z][r]   -= rho[0][z][r]*deltaez;
}
if (r > 2 && setup->point_type[z][r-1] != DITCH) {
//if (setup->point_type[z][r-1] > HVC)
rho[1][z][r-1] += rho[0][z][r]*deltaer * setup->s2[r] * (double) (r-1) / (double) (r-2);
rho[1][z][r]   -= rho[0][z][r]*deltaer * setup->s2[r];
}

//-----------------------------------------------------------
}
}
for (r=1; r<R; r++) {
for (z=1; z<L-2; z++) {
  if(z==4 && r==301){
    printf("In CPU rho[2][%d][%d] is %f\n",z,r, rho[2][z][r]);
  }
if (rho[1][z][r] < 1.0e-14) {
rho[2][z][r] += rho[1][z][r];
continue;
}
// need to r-calculate all the fields
// calc E in r-direction
if (r == 1) {  // r = 0; symmetry implies E_r = 0
E_r = 0;
} 
else if (setup->point_type[z][r] == CONTACT_EDGE) {
E_r = ((v[new_var][z][r] - v[new_var][z][r+1])*setup->dr[1][z][r] +
   (v[new_var][z][r-1] - v[new_var][z][r])*setup->dr[0][z][r]) / (0.2*grid);
} 
else if (setup->point_type[z][r] < INSIDE &&
     setup->point_type[z][r-1] == CONTACT_EDGE) {
E_r =  (v[new_var][z][r-1] - v[new_var][z][r]) * setup->dr[1][z][r-1] / ( 0.1*grid) ;
} 
else if (setup->point_type[z][r] < INSIDE &&
     setup->point_type[z][r+1] == CONTACT_EDGE) {
E_r =  (v[new_var][z][r] - v[new_var][z][r+1]) * setup->dr[0][z][r+1] / ( 0.1*grid) ;
} 
else if (r == R-1) {
E_r = (v[new_var][z][r-1] - v[new_var][z][r])/(0.1*grid);
} 
else {
E_r = (v[new_var][z][r-1] - v[new_var][z][r+1])/(0.2*grid);
}
// calc E in z-direction
// enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
if (setup->point_type[z][r] == CONTACT_EDGE) {
E_z = ((v[new_var][z][r] - v[new_var][z+1][r])*setup->dz[1][z][r] +
   (v[new_var][z-1][r] - v[new_var][z][r])*setup->dz[0][z][r]) / (0.2*grid);
} 
else if (setup->point_type[z][r] < INSIDE &&
     setup->point_type[z-1][r] == CONTACT_EDGE) {
E_z =  (v[new_var][z-1][r] - v[new_var][z][r]) * setup->dz[1][z-1][r] / ( 0.1*grid) ;
} 
else if (setup->point_type[z][r] < INSIDE &&
     setup->point_type[z+1][r] == CONTACT_EDGE) {
E_z =  (v[new_var][z][r] - v[new_var][z+1][r]) * setup->dz[0][z+1][r] / ( 0.1*grid) ;
} 
else if (z == 1) {
E_z = (v[new_var][z][r] - v[new_var][z+1][r])/(0.1*grid);
} 
else if (z == L-1) {
E_z = (v[new_var][z-1][r] - v[new_var][z][r])/(0.1*grid);
} 
else {
E_z = (v[new_var][z-1][r] - v[new_var][z+1][r])/(0.2*grid);
}
ve_z = ve_r = 0;
E = fabs(E_z);
if (E > 1.0) {
for (i=0; E > drift_E[i+1]; i++);
ve_z = fq * (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
}
E = fabs(E_r);
if (E > 1.0) {
for (i=0; E > drift_E[i+1]; i++);
ve_r = fq * (drift_offset[i] + drift_slope[i]*(E - drift_E[i]));
}
/* reduce drift speed at passivated surfaces by a factor of surface_drift_vel_factor */
if (1 &&
((r == idid && z < idd) ||
(r < idid  && z == 1 ) ||
(r >= idid && r <= idod && z == idd))) {
ve_r *= setup->surface_drift_vel_factor;
ve_z *= setup->surface_drift_vel_factor * grid/0.002;  // assume 2-micron-thick roughness/passivation in z
}


//-----------------------------------------------------------

/* do drift to neighboring pixels */
// enum point_types{PC, HVC, INSIDE, PASSIVE, PINCHOFF, DITCH, DITCH_EDGE, CONTACT_EDGE};
if (E_r > 0) {
dre = -TSTEP*ve_r;
} 
else {
dre =  TSTEP*ve_r;
}
if (E_z > 0) {
dze = -TSTEP*ve_z;
} 
else {
dze =  TSTEP*ve_z;
}

if (dre == 0.0) {
i = r;
fr = 1.0;
} 
else {
i = (double) r + dre;
fr = ceil(dre) - dre;
}
if (i<1) {
i = 1;
fr = 1.0;
}
if (i>R-1) {
i = R-1;
fr = 0.0;
}
if (dre > 0 && z < idd && r <= idid && i >= idid) { // ditch ID
i = idid;
fr = 1.0;
}
if (dre < 0 && z < idd && r >= idod && i <= idod) { // ditch OD
i = idod;
fr = 0.0;
}

if (dze == 0.0) {
k = z;
fz = 1.0;
} 
else {
k = (double) z + dze;
fz = ceil(dze) - dze;
}
if (k<1) {
k = 1;
fz = 1.0;
}
if (k>L-1) {
k = L-1;
fz = 0.0;
}
if (dze < 0 && r > idid && r < idod && k < idd) { // ditch depth
k   = idd;
fr  = 1.0;
}
if (0 && r == 100 && z == 10)
printf("r z: %d %d; E_r i dre: %f %d %f; fr = %f\n"
   "r z: %d %d; E_z k dze: %f %d %f; fz = %f\n",
   r, z, E_r, i, dre, fr, r, z, E_z, k, dze, fz);

  if(z==4 && r==301){
  printf("In CPU rho[2][%d][%d] is %f\n",z,r, rho[2][z][r]);
}



// if (i>=1 && i<R && k>=1 && k<L) {
//   if (i > 1 && r > 1) {

//     if(z==4 && r==301){
//       printf("CP 1 In CPU rho[2][%d][%d] is %f\n",z,r, rho[2][z][r]);
//     }
//     rho[2][k  ][i  ] += rho[1][z][r] * fr      *fz       * (double) (r-1) / (double) (i-1);
//     rho[2][k  ][i+1] += rho[1][z][r] * (1.0-fr)*fz       * (double) (r-1) / (double) (i);
//     rho[2][k+1][i  ] += rho[1][z][r] * fr      *(1.0-fz) * (double) (r-1) / (double) (i-1);
//     rho[2][k+1][i+1] += rho[1][z][r] * (1.0-fr)*(1.0-fz) * (double) (r-1) / (double) (i);


//     if(z==4 && r==301){
//       printf("CP 2 In CPU rho[2][%d][%d] is %f\n",z,r, rho[2][z][r]);
//     }


//   } 
//   else if (i > 1) {  // r == 0
//     rho[2][k  ][i  ] += rho[1][z][r] * fr      *fz       / (double) (8*i-8);
//     rho[2][k  ][i+1] += rho[1][z][r] * (1.0-fr)*fz       / (double) (8*i);
//     rho[2][k+1][i  ] += rho[1][z][r] * fr      *(1.0-fz) / (double) (8*i-8);
//     rho[2][k+1][i+1] += rho[1][z][r] * (1.0-fr)*(1.0-fz) / (double) (8*i);
//   } 
//   else if (r > 1) {  // i == 0
//     rho[2][k  ][i  ] += rho[1][z][r] * fr      *fz       * (double) (8*(R+1)-8);
//     rho[2][k  ][i+1] += rho[1][z][r] * (1.0-fr)*fz       * (double) (r-1);
//     rho[2][k+1][i  ] += rho[1][z][r] * fr      *(1.0-fz) * (double) (8*(R+1)-8);
//     rho[2][k+1][i+1] += rho[1][z][r] * (1.0-fr)*(1.0-fz) * (double) (r-1);
//   } 
//   else {             // r == i == 0
//     rho[2][k  ][i  ] += rho[1][z][r] * fr      *fz;
//     rho[2][k  ][i+1] += rho[1][z][r] * (1.0-fr)*fz       / 8.0; // vol_0 / vol_1 = 1/8
//     rho[2][k+1][i  ] += rho[1][z][r] * fr      *(1.0-fz);
//     rho[2][k+1][i+1] += rho[1][z][r] * (1.0-fr)*(1.0-fz) / 8.0;
//   }
//   }

}
}

for (z=0; z<L; z++) {
  for (r=0; r<R; r++) {
    if (setup->point_type[z][r] <= HVC) {
      *gone += rho[2][z][r] * r;
      rho[2][z][r] = 0;
    }
  }
}

return 0;
}